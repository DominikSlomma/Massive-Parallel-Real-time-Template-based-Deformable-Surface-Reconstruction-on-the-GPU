#include "hip/hip_runtime.h"
#include "optim_kernelStreams.cuh"
#include <chrono>
#include <fstream>
#include <sstream>
#include <string>
#include <hipsolver.h>



namespace optim {
    __host__ optGPU::optGPU(int max_iteration, std::vector<Eigen::Vector3d> &reference, std::vector<Eigen::Vector3i> &triangles, Eigen::Matrix3d K, bool verbose)
    :max_iteration_(max_iteration) {
        number_vertices_ = reference.size();
        number_triangles_ = triangles.size();

            // std::cout << triangles[0] << " " << triangles[1] << " " << triangles[2] << std::endl;
            // exit(1);
        
        // W_aux is an auxallary matrix to identify the number of non zero elements and to create the compressed row sparse matrix (csr) for the lower triangle! 
        W_aux = (char*)malloc(number_vertices_ * number_vertices_ * sizeof(char));
        memset( W_aux, 0, number_vertices_ * number_vertices_ * sizeof(char));
        int nnz = 0;

        for(int i=0; i < number_triangles_; i++) {
            Eigen::Vector3i tri = triangles[i]; 
            for(int m=0; m<3;m++) {
                int v1 = tri[m];
                for (int n=m; n<3;n++) {
                    int v2 = tri[n];
                    if (v1 >= v2) {
                        if (W_aux[v2*number_vertices_+v1] == 0)
                            nnz++; 
                        W_aux[v2*number_vertices_+v1] = 1;
                    } else {

                        if (W_aux[v1*number_vertices_+v2] == 0)
                            nnz++;
                        W_aux[v1*number_vertices_+v2] = 1;
                    }           
                }
            }
        }

        nnz_ = nnz;


        // for (int i=0; i< 4; i++) {
        //     for(int j=0; j< 4; j++) {

        //         std::cout << (int)W_aux[i*4 +j] << "\t";

        //     }
        //     std::cout << "\n";

        // }

        

        cholmod_start(&m_cS);


        m_cholSparseE = cholmod_zeros( number_vertices_, 1, CHOLMOD_REAL, &m_cS);
        m_cholSparseS = cholmod_allocate_sparse(number_vertices_,number_vertices_,nnz_,true,true,1,CHOLMOD_REAL,&m_cS); 

        Ap_  = (int*)malloc((number_vertices_ + 1)*sizeof(int));
	    Aii_ = (int*)malloc(nnz*sizeof(int));
        h_AWA = (double*)malloc(number_vertices_ * number_vertices_ *sizeof(double));
        h_AWb = (double*)malloc(number_vertices_ *sizeof(double));

        int *Si = (int *)m_cholSparseS->i;
        int *Sp = (int *)m_cholSparseS->p;



        int* Cp = Ap_;
        int* Ci = Aii_;
        int ii, jj;
        int m = number_vertices_, nZ = 0;
        for ( ii = 0; ii < m; ii++ ) 
        {
            *Cp = nZ;
            *Sp = nZ;
            for( jj=0; jj<=ii; jj++ )
            {
                if ((int)W_aux[jj*m+ii]==1)
                {
                    *Ci++ = jj;
                    *Si++ = jj;
                    nZ++;

                } 
            }

            Cp++;
            Sp++;
        }
        *Cp=nZ;
        *Sp=nZ;



        Eigen::Matrix<int, Eigen::Dynamic, 1> scalarPermutation, blockPermutation;
            
            if (blockPermutation.size() == 0)
				blockPermutation.resize(number_vertices_);
            


                cholmod_sparse auxCholmodSparse;
                auxCholmodSparse.nzmax = nnz_; // Maximale Anzahl der Nicht-Null-Elemente
                auxCholmodSparse.nrow = auxCholmodSparse.ncol = number_vertices_;
                auxCholmodSparse.p = Ap_;   // Spaltenzeiger
                auxCholmodSparse.i = Aii_;   // Zeilenindizes
                auxCholmodSparse.nz = 0;      // Setze auf 0 für CHOLMOD_PATTERN
                auxCholmodSparse.x = nullptr; // Kein Wert-Array benötigt
                auxCholmodSparse.z = nullptr;
                auxCholmodSparse.stype = 1;   // Matrix ist symmetrisch
                auxCholmodSparse.xtype = CHOLMOD_PATTERN;
                auxCholmodSparse.itype = CHOLMOD_INT;
                auxCholmodSparse.dtype = CHOLMOD_DOUBLE;
                auxCholmodSparse.sorted = 1;
                auxCholmodSparse.packed = 1;

                cholmod_print_sparse(&auxCholmodSparse, "aux", &m_cS);

            int amdStatus = cholmod_amd(&auxCholmodSparse, nullptr, 0, blockPermutation.data(), &m_cS);
			if (! amdStatus) {
				std::cout << "AMD error:\n";
                exit(1);
			}

            if (scalarPermutation.size() == 0)
				scalarPermutation.resize(m_cholSparseS->ncol);
			size_t scalarIdx = 0;
			int a = 0;
            
			for ( int i = 0; i < number_vertices_; ++i)
			{
				const int &pp = blockPermutation(i);
				int base = pp*1;
                // std::cout << pp << std::endl;
				// int nCols= (pp==0) ? 6 : 6;

                // int base =  pp*6-1;

				int nCols= 1;

				for ( int j = 0; j < nCols; ++j)
					scalarPermutation(scalarIdx++) = base++;

			}
    
            assert(scalarIdx == m_cholSparseS->ncol);

			// apply the ordering
			m_cS.nmethods = 1 ;
			m_cS.method[0].ordering = CHOLMOD_GIVEN;
            // std::cout << scalarPermutation << std::endl;

            // cholmod_print_sparse(&auxCholmodSparse, "aux", &m_cS);
            cholmod_print_sparse(&auxCholmodSparse, "aux", &m_cS);
            cholmod_print_dense(m_cholSparseE, "back", &m_cS);
			m_cholFactorS = cholmod_analyze_p(m_cholSparseS, scalarPermutation.data(), NULL, 0, &m_cS);

        // Si[0] = 0;
        // Si[1] = 0;
        // Si[2] = 1;
        // Si[3] = 0;
        // Si[4] = 1;
        // Si[5] = 2;
        // Si[6] = 1;
        // Si[7] = 2;
        // Si[8] = 3;


        // Sp[0] = 0;
        // Sp[1] = 1;
        // Sp[2] = 3;
        // Sp[3] = 6;
        // Sp[4] = 9;

        // exit(1);


        int counter = 0;
        rowPtrCsr.push_back(counter);

        for (int i=0; i<number_vertices_; i++) {

            for(int j=0;j<number_vertices_;j++) {
                int idx = (int)W_aux[i*number_vertices_+j];
                if(idx == 1) {
                    colPtrCsr.push_back(j);
                    counter++;
                    valPtrCsr.push_back(0);
                }
                // std::cout << (int)W_aux[i*number_vertices_+j] << "\t";
            }
            rowPtrCsr.push_back(counter);

            // std::cout << std::endl;
        }
        // nnz_ = nnz;

        // std::cout <<  nnz << std::endl;

        // for(int i=0; i<colPtrCsr.size();i++) {
        //     std::cout << colPtrCsr[i] << "\t";
        // }
        // std::cout << std::endl;

        // for(int i=0; i<rowPtrCsr.size();i++) {
        //     std::cout << rowPtrCsr[i] << "\t";
        // }
        // std::cout << std::endl;

        // for(int i=0; i<valPtrCsr.size();i++) {
        //     std::cout << valPtrCsr[i] << "\t";
        // }
        // std::cout << std::endl;
        // exit(1);
        
        // reference_ = (double*)malloc(number_vertices_*3*sizeof(double));
        // vertices_ = (double*)malloc(number_vertices_*3*sizeof(double)); // ist ok
        
        hipError_t err1 = hipMalloc((void**)&d_vertices_, number_vertices_*3*sizeof(double));
        hipError_t err2 = hipMalloc((void**)&d_reference_, number_vertices_*3*sizeof(double));
        hipError_t err3 = hipMalloc((void**)&d_triangles_, number_triangles_*3*sizeof(int));
        hipError_t err4 = hipMalloc((void**)&d_K_, 4*sizeof(double));
        hipError_t err5 = hipMalloc((void**)&d_max_iteration_, 1*sizeof(int));
        // hipError_t err6 = hipMalloc((void**)&d_break_criteria, 1*sizeof(double));
        hipError_t err7 = hipMalloc((void**)&d_obs_, number_vertices_*2*sizeof(double));

        // hipError_t err4 = hipMalloc((void**)&d_sub_vert_, number_triangles_*3*3*sizeof(double));
        hipError_t err8 = hipMalloc((void**)&d_cost_, number_triangles_*20*sizeof(double));
        hipError_t err9 = hipMalloc((void**)&d_dx_, number_triangles_*max_iteration*sizeof(double));

        hipError_t err10 = hipMalloc((void**)&d_number_triangles_, 1*sizeof(int));
        hipError_t err11 = hipMalloc((void**)&d_number_vertices_, 1*sizeof(int));
        hipError_t err12 = hipMalloc((void**)&d_number_observation_, 1*sizeof(int));
        hipError_t err13 = hipMalloc((void**)&d_bar_, number_triangles_*3*sizeof(double));


        hipError_t err14 = hipMalloc((void**)&d_colPtrCsr, colPtrCsr.size()*sizeof(int));
        hipError_t err15 = hipMalloc((void**)&d_rowPtrCsr, rowPtrCsr.size()*sizeof(int));
        hipError_t err16 = hipMalloc((void**)&d_valPtrCsr, valPtrCsr.size()*sizeof(double));
        hipError_t err17 = hipMalloc((void**)&d_b, number_vertices_*sizeof(double));
        hipError_t err18 = hipMalloc((void**)&d_x, number_vertices_*sizeof(double));
        hipError_t err19 = hipMalloc((void**)&d_information_matrix, number_triangles_*9*sizeof(double));
        hipError_t err20 = hipMalloc((void**)&d_nnz, 1*sizeof(int));
        hipError_t err21 = hipMalloc((void**)&d_d, number_triangles_*3*sizeof(double));

        hipError_t err22 = hipMalloc((void**)&d_c_unit, number_triangles_*3*sizeof(double));
        hipError_t err23 = hipMalloc((void**)&d_determinant, number_triangles_*6*sizeof(double));
        hipError_t err24 = hipMalloc((void**)&d_g, number_triangles_*3*sizeof(double));
        hipError_t err25 = hipMalloc((void**)&d_H, number_triangles_*9*sizeof(double));
        hipError_t err26 = hipMalloc((void**)&d_AWA, number_vertices_*number_vertices_*sizeof(double));
        hipError_t err27 = hipMalloc((void**)&d_AWb, number_vertices_*sizeof(double));



        if ((err1 != hipSuccess) || (err2 != hipSuccess) || (err3 != hipSuccess) || (err4 != hipSuccess)
        || (err5 != hipSuccess) || (err7 != hipSuccess) || (err8 != hipSuccess) 
        || (err9 != hipSuccess) || (err10 != hipSuccess) || (err11 != hipSuccess) || (err12 != hipSuccess)
        || (err13 != hipSuccess) || (err14 != hipSuccess) || (err15 != hipSuccess) || (err16 != hipSuccess) 
        || (err17 != hipSuccess) || (err18 != hipSuccess) || (err19 != hipSuccess) || (err20 != hipSuccess)
        || (err21 != hipSuccess) || (err22 != hipSuccess) || (err23 != hipSuccess) || (err24 != hipSuccess)
        || (err25 != hipSuccess)) {
            printf("CUDA Error1: %s\n", hipGetErrorString(err1));
            printf("CUDA Error2: %s\n", hipGetErrorString(err2));
            printf("CUDA Error3: %s\n", hipGetErrorString(err3));
            printf("CUDA Error4: %s\n", hipGetErrorString(err4));
            printf("CUDA Error5: %s\n", hipGetErrorString(err5));
            printf("CUDA Error7: %s\n", hipGetErrorString(err7));
            printf("CUDA Error8: %s\n", hipGetErrorString(err8));
            printf("CUDA Error9: %s\n", hipGetErrorString(err9));
            printf("CUDA Error10: %s\n", hipGetErrorString(err10));
            printf("CUDA Error11: %s\n", hipGetErrorString(err11));
            printf("CUDA Error12: %s\n", hipGetErrorString(err12));
            printf("CUDA Error13: %s\n", hipGetErrorString(err13));
            printf("CUDA Error14: %s\n", hipGetErrorString(err14));
            printf("CUDA Error15: %s\n", hipGetErrorString(err15));
            printf("CUDA Error16: %s\n", hipGetErrorString(err16));
            printf("CUDA Error17: %s\n", hipGetErrorString(err17));
            printf("CUDA Error18: %s\n", hipGetErrorString(err18));
            printf("CUDA Error19: %s\n", hipGetErrorString(err19));
            printf("CUDA Error20: %s\n", hipGetErrorString(err20));
            printf("CUDA Error21: %s\n", hipGetErrorString(err21));
            printf("CUDA Error22: %s\n", hipGetErrorString(err22));
            printf("CUDA Error23: %s\n", hipGetErrorString(err23));
            printf("CUDA Error24: %s\n", hipGetErrorString(err24));
            printf("CUDA Error25: %s\n", hipGetErrorString(err25));
            // Handle the error, e.g., by exiting the function
            exit(1);
        }
        
        

        double K_tmp[4];
        K_tmp[0] = K(0,0);
        K_tmp[1] = K(0,2);
        K_tmp[2] = K(1,1);
        K_tmp[3] = K(1,2);
        
        hipMemcpy(d_triangles_, triangles.data(), number_triangles_*3*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_reference_, reference.data(), number_vertices_*3*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_K_, K_tmp, 4*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_max_iteration_, &max_iteration, 1*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_number_triangles_, &number_triangles_, 1*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_number_vertices_, &number_vertices_, 1*sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_AWA, 0, number_vertices_ * number_vertices_ * sizeof(double));
        hipMemset(d_AWb, 0, number_vertices_  * sizeof(double));
        // hipMemcpy(d_rowPtrCsr, rowPtrCsr.data(), rowPtrCsr.size() * sizeof(int), hipMemcpyHostToDevice);
        // hipMemcpy(d_colPtrCsr, colPtrCsr.data(), colPtrCsr.size() * sizeof(int), hipMemcpyHostToDevice);
        // hipMemcpy(d_valPtrCsr, valPtrCsr.data(), nnz * sizeof(double), hipMemcpyHostToDevice);
        // hipMemcpy(d_nnz, &nnz, 1 * sizeof(int), hipMemcpyHostToDevice);

        // std::cout << rowPtrCsr.size() << std::endl; exit(1);

        // hipMemcpy(d_number_observation_, &d_number_observation_, 1*sizeof(int), hipMemcpyHostToDevice);
        // hipMemcpy(d_break_criteria, break_criteria, 1*sizeof(double), hipMemcpyHostToDevice);

        // Todo: Cuda informationen ausprinten wenn gefordert!

        // Todo: Compute \bar{d_{ij}} hier!
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }


        for(int i = 0; i<triangles.size();i++) {
            Eigen::Vector3i tri = triangles[i];

            if ((tri.x() < 0 || tri.x() >= reference.size()) || (tri.y() < 0 || tri.y() >= reference.size()) || (tri.z() < 0 || tri.z() >= reference.size())) {
                printf("shit %d, \n", i);
            }
        }
        // std::cout << number_triangles_ << std::endl;


        hipStream_t stream1, stream2, stream3;
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);

        int numBlocks = 1;
        int numThreads = number_triangles_;
        if(numThreads > 1024) {
            int newBlocksize = std::ceil(numThreads / 1024);
            numThreads = std::ceil(numThreads / newBlocksize);
            numBlocks = newBlocksize;
        }



        compute_d_bar_x<<<numBlocks, numThreads, 0, stream1>>>(d_number_triangles_, d_triangles_, d_reference_, d_bar_);
        compute_d_bar_y<<<numBlocks, numThreads, 0, stream2>>>(d_number_triangles_, d_triangles_, d_reference_, d_bar_);
        compute_d_bar_z<<<numBlocks, numThreads, 0, stream3>>>(d_number_triangles_, d_triangles_, d_reference_, d_bar_);

        hipDeviceSynchronize();



        // hipStreamSynchronize(stream1);
        // hipStreamSynchronize(stream2);
        // hipStreamSynchronize(stream3);



        // compute_d_bar<<<number_triangles_,3>>>(d_triangles_, d_reference_, d_bar_);
        // hipDeviceSynchronize();

        hipStreamCreate(&stream_cost);
        hipStreamCreate(&stream_G);
        hipStreamCreate(&stream_H);
        hipStreamCreate(&stream_det);
        hipStreamCreate(&stream_adj);
        hipStreamCreate(&stream_up);
        hipStreamCreate(&stream_AWA);
        hipStreamCreate(&stream_AWb);

        hipEventCreate(&event_cost);
        hipEventCreate(&event_G);
        hipEventCreate(&event_H);
        hipEventCreate(&event_det);
        hipEventCreate(&event_adj);
        hipEventCreate(&event_up);
        hipEventCreate(&event_AWA);
        hipEventCreate(&event_AWb);


        int blocksize;
        int threadsize;

        blocksize = 1;
        threadsize = number_triangles_ * 1;

        if(threadsize > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            blocksize = newBlocksize;
            threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
        }

        blocksize_cost = blocksize;
        threadsize_cost = threadsize;


        blocksize = 1;
        threadsize = number_triangles_ * 3;

        if(threadsize > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            blocksize = newBlocksize;
            threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
        }

        blocksize_G = blocksize;
        threadsize_G = threadsize;

        blocksize_H = blocksize;
        threadsize_H = threadsize;

        blocksize = 1;
        threadsize = number_triangles_ * 6;

        if(threadsize > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            blocksize = newBlocksize;
            threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
        }

        blocksize_det = blocksize;
        threadsize_det = threadsize;

        blocksize = 1;
        threadsize = number_triangles_ * 9;
        if(threadsize > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            blocksize = newBlocksize;
            threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
        }


        blocksize_adj = blocksize;
        threadsize_adj = threadsize;


        blocksize = 1;
        threadsize = number_triangles_ * 3;
        if(threadsize > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            blocksize = newBlocksize;
            threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
        }

        blocksize_up = blocksize;
        threadsize_up = threadsize;

        // double depth[6];

        // hipMemcpy(depth, d_bar_, 6*sizeof(double), hipMemcpyDeviceToHost);

        // std::cout << depth[0] << " " << depth[1] << " " << depth[2] << std::endl;
        // std::cout << depth[3] << " " << depth[4] << " " << depth[5] << std::endl;
        // exit(1);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }
        // exit(1);
        // printf("Con\n");
        // Todo: possible to release d_reference!


        // status_ = hipsolverSpCreate(&cusolverH_); 
        // checkCusolverStatus(status_);


        // hipsparseCreateMatDescr(&descrA_);
        // hipsparseSetMatType(descrA_, HIPSPARSE_MATRIX_TYPE_GENERAL);
        // hipsparseSetMatIndexBase(descrA_, HIPSPARSE_INDEX_BASE_ZERO);
    }

    __host__ optGPU::~optGPU(){
        // hipFree(d_vertices_);
        // hipFree(d_reference_);
        // hipFree(d_bar_);
        // hipFree(d_K_);
        // hipFree(d_obs_);
        // hipFree(d_cost_);
        // hipFree(d_dx_);
        // hipFree(d_max_iteration_);
        // hipFree(d_triangles_);
        // hipFree(d_number_triangles_);
        // hipFree(d_number_vertices_);
        // hipFree(d_number_observation_);
    }


    void optGPU::checkCusolverStatus(hipsolverStatus_t status) {
        switch (status) {
            case HIPSOLVER_STATUS_SUCCESS:
                std::cout << "Operation completed successfully." << std::endl;
                break;
            case HIPSOLVER_STATUS_NOT_INITIALIZED:
                std::cout << "HIPSOLVER_STATUS_NOT_INITIALIZED: The library was not initialized." << std::endl;
                break;
            case HIPSOLVER_STATUS_ALLOC_FAILED:
                std::cout << "HIPSOLVER_STATUS_ALLOC_FAILED: The resources could not be allocated." << std::endl;
                break;
            case HIPSOLVER_STATUS_INVALID_VALUE:
                std::cout << "HIPSOLVER_STATUS_INVALID_VALUE: Invalid parameters were passed." << std::endl;
                break;
            case HIPSOLVER_STATUS_ARCH_MISMATCH:
                std::cout << "HIPSOLVER_STATUS_ARCH_MISMATCH: The device only supports compute capability 5.0 and above." << std::endl;
                break;
            case HIPSOLVER_STATUS_INTERNAL_ERROR:
                std::cout << "HIPSOLVER_STATUS_INTERNAL_ERROR: An internal operation failed." << std::endl;
                break;
            case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
                std::cout << "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: The matrix type is not supported." << std::endl;
                break;
            default:
                std::cout << "Unknown error occurred." << std::endl;
        }
    }


    // __host__ void optGPU::setParamater(double *observation, std::unordered_map<int,int> &unordered_mapping_vertices, std::unordered_map<int,int> &unordered_mapping_triangles, int number_vertices, int number_triangles, int number_observation)

    __host__ void optGPU::setParamater(std::vector<double> observation, int number_observation) {

        // hipMemset(d_AWA, 0, number_vertices_ * number_vertices_ * sizeof(float));
        // hipMemset(d_AWb, 0, number_vertices_  * sizeof(float));
        // std::cout << "hier\n"; 
        
        int num_obs[1];
        *num_obs = number_observation; 
        // std::cout << "hier\n";
        hipMemcpy(d_obs_, observation.data(), number_vertices_*2*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_number_observation_, num_obs, 1*sizeof(int), hipMemcpyHostToDevice); 

        // auto start1 = std::chrono::high_resolution_clock::now();

        // std::cout << "hier\n"; 

        int numBlock = 1;
        int numthread = number_observation;


        if (numthread> 1024) {
            int newBlocksize = std::ceil(static_cast<float>(numthread)/1024);
            numBlock = newBlocksize;
            numthread = std::ceil(static_cast<float>(numthread) / newBlocksize);
        }

        obs2unitvector<<<numBlock,numthread>>>(d_obs_, d_vertices_, d_K_, d_number_observation_);

        numBlock = 1;
        numthread = number_triangles_*3;

        if (numthread > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(numthread)/1024);
            numBlock = newBlocksize;
            numthread = std::ceil(static_cast<float>(numthread) / newBlocksize);
        }
        hipDeviceSynchronize();

        // std::cout << numBlock << " " << numthread << std::endl; exit(1); 
        compute_constantUnits<<<numBlock, numthread>>>(d_c_unit, d_triangles_, d_number_triangles_, d_vertices_);
        hipDeviceSynchronize();
        // std::cout << number_triangles_ << " "<< number_vertices_ << std::endl; 
        // exit(1);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
            exit(1);
        }



        numBlock = 1;
        numthread = number_triangles_ * 3;

        if (numthread > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(numthread)/1024);
            numBlock = newBlocksize;
            numthread = std::ceil(static_cast<float>(numthread) / newBlocksize);
        }


        compute_d<<<numBlock, numthread>>>(d_d, d_reference_, d_number_triangles_, d_triangles_);



        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err));
        }

        // std::cout << "hier\n"; 
        
        // auto end1 = std::chrono::high_resolution_clock::now();
        // // Berechnung der verstrichenen Zeit
        // auto elapsed1 = std::chrono::duration_cast<std::chrono::microseconds>(end1 - start1);

        // // Ausgabe der verstrichenen Zeit in Sekunden
        // std::cout <<" Verstrichene Zeit: " << elapsed1.count() << " Mikrosekunden" << std::endl;


    }


    __host__ void optGPU::run() {


        // int blocksize;
        // int threadsize;

        // blocksize = 1;
        // threadsize = number_triangles_ * 1;

        // if(threadsize > 1024) {
        //     int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
        //     blocksize = newBlocksize;
        //     threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
        // }

        // hipEvent_t start_, stop_;
        // float elapsedTime;

        // hipEventCreate(&start_);
        // hipEventCreate(&stop_);

        for (int iter = 0; iter < 10; iter++) {
            auto start = std::chrono::high_resolution_clock::now();
            // hipEventRecord(start_, 0);
            
            
            
            compute_cost_new<<<blocksize_cost, threadsize_cost, 0, stream_cost>>>(d_bar_, d_triangles_, d_number_triangles_, d_d, d_vertices_, d_cost_);
            hipError_t err4 = hipGetLastError();
        if (err4 != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err4));
        }
            hipEventRecord(event_cost, stream_cost);
         
         hipError_t err3 = hipGetLastError();
        if (err3 != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err3));
        }
            // hipDeviceSynchronize();

            // blocksize = 1;
            // threadsize = number_triangles_ * 3;

            // if(threadsize > 1024) {
            //     int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            //     blocksize = newBlocksize;
            //     threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
            // }
            
            computeG_new<<<blocksize_G, threadsize_G, 0, stream_G>>>(d_g, d_triangles_, d_vertices_, d_d, d_bar_, d_c_unit, d_number_triangles_);
            hipEventRecord(event_G, stream_G);



            computeH_new<<<blocksize_H, threadsize_H, 0, stream_H>>>(d_H, d_triangles_, d_vertices_, d_d, d_c_unit, d_number_triangles_);
            hipEventRecord(event_H, stream_H);
            
            
            
            // hipDeviceSynchronize();


            // double A[18] = {1.2, 0.5, 0.3, 0.4, 1.1, 0.2,
            // 0.3, 0.2, 1.3,1.2, 0.5, 0.3, 0.4, 1.1, 0.2,
            // 0.3, 0.2, 1.3};

            // hipMemcpy(d_H, A, 18*sizeof(double), hipMemcpyHostToDevice);

            // blocksize = 1;
            // threadsize = number_triangles_ * 6;

            // if(threadsize > 1024) {
            //     int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            //     blocksize = newBlocksize;
            //     threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
            // }
            hipStreamWaitEvent(stream_det, event_H, 0);
            compute_determinante_new<<<blocksize_det, threadsize_det, 0, stream_det>>>(d_H, d_number_triangles_, d_determinant);
            hipEventRecord(event_det, stream_det);



            // blocksize = 1;
            // threadsize = number_triangles_ * 9;
            // if(threadsize > 1024) {
            //     int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            //     blocksize = newBlocksize;run(wBlocksize);
            // }
            hipStreamWaitEvent(stream_adj, event_H, 0);
            compute_adjugate_new<<<blocksize_adj, threadsize_adj, 0, stream_adj>>>(d_H, d_number_triangles_, d_information_matrix);
            hipEventRecord(event_adj, stream_adj);

            
            
            // hipDeviceSynchronize();

            // blocksize = 1;
            // threadsize = number_triangles_ * 3;
            // if(threadsize > 1024) {
            //     int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            //     blocksize = newBlocksize;
            //     threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
            // }

            hipStreamWaitEvent(stream_up, event_det, 0);
            hipStreamWaitEvent(stream_up, event_adj, 0);
            hipStreamWaitEvent(stream_up, event_G, 0);

            update_dx<<<blocksize_up, threadsize_up, 0, stream_up>>>(d_d, d_number_triangles_, d_information_matrix, d_determinant, d_g, d_cost_);
            hipEventRecord(event_up, stream_up);


            hipStreamSynchronize(stream_up);

            // hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("CUDA Error: %s\n", hipGetErrorString(err));
            }
            auto end = std::chrono::high_resolution_clock::now();
            auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
            // hipEventRecord(stop_, 0);
            // hipEventSynchronize(stop_);
            // hipEventElapsedTime(&elapsedTime, start_, stop_);
            // hipDeviceSynchronize();
            // hipStreamSynchronize(stream_cost);

            double cost1[20];
            // cost1 = (double*)malloc(1*20*sizeof(double));
            hipMemcpy(cost1, d_cost_, 1*20*sizeof(double), hipMemcpyDeviceToHost);
            std::cout << "Itertation: " << iter <<" Error: " << cost1[0] << " Time: " << elapsed.count() << " Total time: " << 0 << std::endl;
            // std::cout << cost1[1] << " " << cost1[2] << " " << cost1[3] << std::endl;


        }
        hipError_t err1 = hipGetLastError();
        if (err1 != hipSuccess) {
            printf("CUDA Error: %s\n", hipGetErrorString(err1));
        }
        // exit(1);

       

        int blocksize;
        int threadsize;

        blocksize = 1;
        threadsize = 3*number_triangles_;

        if(threadsize > 1024) {
            int newBlocksize = std::ceil(static_cast<float>(threadsize)/1024);
            blocksize = newBlocksize;
            threadsize = std::ceil(static_cast<float>(threadsize) / newBlocksize);
        }


        compute_AWA<<<blocksize, threadsize, 0, stream_AWA>>>(d_H, d_triangles_, d_number_triangles_, d_number_vertices_, d_AWA);
        hipEventRecord(event_AWA, stream_AWA);

        compute_AWB<<<blocksize, threadsize, 0, stream_AWb>>>(d_H, d_d, d_triangles_, d_number_triangles_, d_number_vertices_, d_AWb);
        hipEventRecord(event_AWb, stream_AWb);


        hipStreamSynchronize(stream_AWb);
        hipStreamSynchronize(stream_AWA);

        // double hhh[16];
        // double hhhh[4];
        hipMemcpy(h_AWA, d_AWA, number_vertices_*number_vertices_*sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_AWb, d_AWb, number_vertices_*sizeof(double), hipMemcpyDeviceToHost);


        // int num_rows = rowPtrCsr.size() - 1;  // Anzahl der Zeilen
        // for (int row = 0; row < num_rows; ++row) {
        //     // Beginne und ende für die aktuelle Zeile im col_idx Array
        //     int start = rowPtrCsr[row];
        //     int end = rowPtrCsr[row + 1];

        //     // Iteriere über die Spaltenindizes dieser Zeile
        //     for (int idx = start; idx < end; ++idx) {
        //         int col = colPtrCsr[idx];  // Spaltenindex des aktuellen Elements
        //         std::cout << "Row: " << row << ", Col: " << col << std::endl;
        //     }
        // }

        // int num_rows = rowPtrCsr.size() - 1;
        // for (int row=0; row < num_rows; row++) {
        //     int start = rowPtrCsr[row];
        //     int end = rowPtrCsr[row + 1];

        //     for(int idx = start; idx < end; idx++) {
        //         int col = colPtrCsr[idx];
        //         valPtrCsr[idx] = hhh[row * number_vertices_ + col];
        //     }
        // }

        // for(int i=0; i< 9; i++) {
        //     std::cout << Aii_[i] << std::endl;
        // }
        // std::cout << std::endl;

        // for(int i=0; i< 5; i++) {
        //     std::cout << Ap_[i] << std::endl;
        // }
        // std::cout << std::endl;


        // for(int i=0; i< valPtrCsr.size(); i++) {
        //     std::cout << valPtrCsr[i] << std::endl;
        // }
        // std::cout << std::endl;

        

        

         
            // cholmod_print_sparse(m_cholSparseS, "S", &m_cS);

        // for(int i=0; i< rowPtrCsr.size();i++) {
        //     Ai[i] = rowPtrCsr[i];
        // }

        // Ai[0] = 0;
        // Ai[1] = 0;
        // Ai[2] = 1;

        // Ai[3] = 0;
        // Ai[4] = 1;
        // Ai[5] = 2;

        // Ai[6] = 1;
        // Ai[7] = 2;
        // Ai[8] = 3;

        // int Aii[9];
        // Aii[0] = 0;
        // Aii[1] = 0;
        // Aii[2] = 1;
        // Aii[3] = 0;
        // Aii[4] = 1;
        // Aii[5] = 2;
        // Aii[6] = 1;
        // Aii[7] = 2;
        // Aii[8] = 3;

        // for(int i=0; i< colPtrCsr.size();i++) {
        //     Ap[i] = colPtrCsr[i];
        // }

        // Ap[0] = 0;
        // Ap[1] = 1;
        // Ap[2] = 3;
        // Ap[3] = 6;
        // Ap[4] = 9;

        // int App[5];
        // App[0] = 0;
        // App[1] = 1;
        // App[2] = 3;
        // App[3] = 6;
        // App[4] = 9;


        double *Sx = (double *)m_cholSparseS->x;
        int *Sp = (int *) m_cholSparseS->p;
        int *Si = (int *) m_cholSparseS->i;

        for(int col_id = 0; col_id < number_vertices_; col_id++) {

            int start = Sp[col_id];
            int end = Sp[col_id+1];
            for(int idx = start; idx < end; idx++) {
                int row = Si[idx];
                Sx[idx] = h_AWA[row * number_vertices_ + col_id];
                // std::cout <<  col_id << " " << row << " " << Sx[idx] << std::endl;
            }
        }


        // for(int i=0; i< valPtrCsr.size();i++) {
        //     Sx[i] = i+1; // valPtrCsr[i];
        // }

            // cholmod_print_sparse(m_cholSparseS, "S", &m_cS);

        double *Ex = (double*)m_cholSparseE->x;

        for(int ii = 0; ii < number_vertices_; ii++) {
            Ex[ii] = h_AWb[ii];//hhhh[ii];

            // std::cout << Ex[ii] << std::endl;
        }

       
    // int *testi = App;
    // int *testp = Aii;

 
            // Eigen::Matrix<int, Eigen::Dynamic, 1> scalarPermutation, blockPermutation;
            
            // if (blockPermutation.size() == 0)
			// 	blockPermutation.resize(number_vertices_);
            


            //     cholmod_sparse auxCholmodSparse;
            //     auxCholmodSparse.nzmax = nnz_; // Maximale Anzahl der Nicht-Null-Elemente
            //     auxCholmodSparse.nrow = auxCholmodSparse.ncol = number_vertices_;
            //     auxCholmodSparse.p = Ap_;   // Spaltenzeiger
            //     auxCholmodSparse.i = Aii_;   // Zeilenindizes
            //     auxCholmodSparse.nz = 0;      // Setze auf 0 für CHOLMOD_PATTERN
            //     auxCholmodSparse.x = nullptr; // Kein Wert-Array benötigt
            //     auxCholmodSparse.z = nullptr;
            //     auxCholmodSparse.stype = 1;   // Matrix ist symmetrisch
            //     auxCholmodSparse.xtype = CHOLMOD_PATTERN;
            //     auxCholmodSparse.itype = CHOLMOD_INT;
            //     auxCholmodSparse.dtype = CHOLMOD_DOUBLE;
            //     auxCholmodSparse.sorted = 1;
            //     auxCholmodSparse.packed = 1;

            //     cholmod_print_sparse(&auxCholmodSparse, "aux", &m_cS);

            // int amdStatus = cholmod_amd(&auxCholmodSparse, nullptr, 0, blockPermutation.data(), &m_cS);
			// if (! amdStatus) {
			// 	std::cout << "AMD error:\n";
            //     exit(1);
			// }

            // if (scalarPermutation.size() == 0)
			// 	scalarPermutation.resize(m_cholSparseS->ncol);
			// size_t scalarIdx = 0;
			// int a = 0;
            
			// for ( int i = 0; i < number_vertices_; ++i)
			// {
			// 	const int &pp = blockPermutation(i);
			// 	int base = pp*1;
            //     // std::cout << pp << std::endl;
			// 	// int nCols= (pp==0) ? 6 : 6;

            //     // int base =  pp*6-1;

			// 	int nCols= 1;

			// 	for ( int j = 0; j < nCols; ++j)
			// 		scalarPermutation(scalarIdx++) = base++;

			// }
    
            // assert(scalarIdx == m_cholSparseS->ncol);

			// // apply the ordering
			// m_cS.nmethods = 1 ;
			// m_cS.method[0].ordering = CHOLMOD_GIVEN;
            // // std::cout << scalarPermutation << std::endl;

            // // cholmod_print_sparse(&auxCholmodSparse, "aux", &m_cS);
            // cholmod_print_sparse(&auxCholmodSparse, "aux", &m_cS);
            // cholmod_print_dense(m_cholSparseE, "back", &m_cS);
			// m_cholFactorS = cholmod_analyze_p(m_cholSparseS, scalarPermutation.data(), NULL, 0, &m_cS);

            // bool init = true;

            // cholmod_print_factor(m_cholFactorS, "L", &m_cS);

            cholmod_factorize(m_cholSparseS, m_cholFactorS, &m_cS); 
            
            m_cholSparseR = cholmod_solve (CHOLMOD_A, m_cholFactorS, m_cholSparseE, &m_cS) ;

            double *rx = (double*)m_cholSparseR->x;


            // for (int ii=0; ii< number_vertices_; ii++) {
            //     std::cout << "d" << ii << ":\t" << rx[ii] << std::endl;
            // }
            // std::cout << std::endl;
        // valPtrCsr[tid] = 0;
        // for(int col_id=0; col_id < nnz_; col_id++) {

        //     if((tid >= rowPtrCsr[i]) && (tid < rowPtrCsr[i+1])) {
        //         rowIdx = i;
        //         // colIdx = colPtrCsr[tid];
        //         break;
        //     }
        // }


        // for(int i=0; i < ; i++) {
        //     int tri1, tri2, tri3;
        //     bool rowExist = false;
        //     bool colExist = false;

        //     int rIdx = -1;
        //     int cIdx = -1;

        //     tri1 = triangles[i*3];
        //     tri2 = triangles[i*3+1];
        //     tri3 = triangles[i*3+2];
            
        //     existIDInW(tri1, tri2, tri3, rowIdx, rowExist, rIdx);
        //     existIDInW(tri1, tri2, tri3, colIdx, colExist, cIdx);

        //     if (rowExist && colExist) {
                
        //         double* info_mat_tmp = info_matrix + (i*9);
        //         valPtrCsr[tid] += info_mat_tmp[rIdx*3+cIdx]; // row or col IDx kann größer als drei sein!
        //     }
        // }

    // hipsolverHandle_t cusolverHandle;
    // hipsolverDnCreate(&cusolverHandle);
    // hipsolverDnParams_t cusolverParams;
    // hipsolverDnCreateParams(&cusolverParams);

    // size_t workspaceInBytesOnDevice;
    // size_t workspaceInBytesOnHost;
    // const int lda = 4;

    // void* bufferOnDevice = nullptr;
    // void* bufferOnHost = nullptr;
    // int* d_info = nullptr;
    // hipMalloc(&d_info, sizeof(int));
    
    // hipsolverDnXpotrf_bufferSize(cusolverHandle, cusolverParams, HIPBLAS_FILL_MODE_UPPER, lda, HIP_R_64F, d_AWA, lda, HIP_R_64F, &workspaceInBytesOnDevice, &workspaceInBytesOnHost);
    // hipMalloc(&bufferOnDevice, workspaceInBytesOnDevice);
    // bufferOnHost = malloc(workspaceInBytesOnHost);
    
    // // Perform the Cholesky decomposition
    // hipsolverDnXpotrf(
    //     cusolverHandle,
    //     cusolverParams,
    //     HIPBLAS_FILL_MODE_UPPER,
    //     lda,
    //     HIP_R_64F,
    //     d_AWA,
    //     lda,
    //     HIP_R_64F,
    //     bufferOnDevice,
    //     workspaceInBytesOnDevice,
    //     bufferOnHost,
    //     workspaceInBytesOnHost,
    //     d_info
    // );


    // hipsolverDnXpotrs(cusolverHandle,
    //     cusolverParams,
    //     HIPBLAS_FILL_MODE_UPPER,
    //     lda,
    //     1, // Number of right-hand sides (1 vector)
    //     HIP_R_64F,
    //     d_AWA,
    //     lda,
    //     HIP_R_64F,
    //     d_AWb,
    //     lda,
    //     d_info
    // );
        // hipsolverDnXpotrf()
        // cusolverDNXpotrf();



// int numBlock = 1;
//         int numthread = number_observation;
//         if (numthread> 1024) {
//             int newBlocksize = std::ceil(static_cast<float>(numthread)/1024);
//             numBlock = newBlocksize;
//             numthread = std::ceil(static_cast<float>(numthread) / newBlocksize);
//         }


        // kernel call
        // Todo: -> use d_bar
        //<<<number_triangles_,18>>>

        // int blocknumber = 1;
        // int threadnumber = static_cast<int>(std::ceil(number_triangles_ * 18 ));
        
        // // 1008 is the max number of threads which can be used to operate in a full block for a complete triangle!
        // if (threadnumber > 1008) {
        //     blocknumber = static_cast<int>(std::ceil(threadnumber/1008)) ;
        //     threadnumber = 1008;
        // }

        // std::cout << threadnumber << " " << blocknumber << std::endl;

        // size_t sharedMemSize = 68 * (threadnumber / 18) * sizeof(double);

        // gauss_newton<<<blocknumber, threadnumber, sharedMemSize>>>(d_triangles_, d_bar_, d_reference_, d_vertices_, d_K_, d_number_triangles_,d_triangles_, d_obs_, d_triangles_, d_max_iteration_, d_cost_, d_information_matrix, d_d);
        // hipDeviceSynchronize();
        
        // // std::cout << "I'm here: \n";
        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) {
        //     printf("CUDA Error: %s\n", hipGetErrorString(err));
        // }
        // int num_threads_total = nnz_ + number_vertices_;
        // int blockNum;
        // int threadNum;
        // if(num_threads_total > 1024) {
        //     if (num_threads_total%2 == 1)
        //         num_threads_total++;
        //     blockNum = num_threads_total/1024+1;
        //     threadNum = num_threads_total/blockNum;
        // } else {
        //     blockNum = 1;
        //     threadNum = num_threads_total;
        // }


        // std::cout << "blocknum: " << blockNum << " threadnum: " << threadNum <<  " nnz: " << nnz_ << " number Vert: " << number_vertices_ << std::endl; 
        // prepareCuSolver<<<blockNum , threadNum>>>(d_number_triangles_, d_number_vertices_, d_triangles_, d_colPtrCsr, d_rowPtrCsr, d_valPtrCsr, d_b, d_information_matrix, d_d, d_nnz); 
        // err = hipGetLastError();       //int* num_triangles, int* m, int* triangles, int* colPtrCsr, int* rowPtrCsr, double* valPtrCsr, double* b, double* info_matrix, double* d, int* nnz, int* number_vertices
        // if (err != hipSuccess) {
        //     printf("CUDA Error: %s\n", hipGetErrorString(err));
        // }

        // status_ = hipsolverSpDcsrlsvchol(cusolverH_, number_vertices_, nnz_, descrA_, d_valPtrCsr, d_rowPtrCsr, d_colPtrCsr, d_b, 0, 0, d_x, &singularity_);
        // checkCusolverStatus(status_);


        // if (singularity_ > 0) {
        //     std::cout << "Warnung: Die Matrix ist singulär bei index " << singularity_ << std::endl;
        // }


    }

    __host__ void optGPU::getVertices() {
        std::cout << "\n Daten\n";
        // Kopiere Daten von der GPU zur CPU
        // double* cost_ = (double*)malloc(number_triangles_*20*sizeof(double));

        // hipMemcpy(cost_, d_cost_, number_triangles_*20*sizeof(double), hipMemcpyDeviceToHost);

        // std::cout << "error norm" << std::endl;
        // for(int i=0;i<20;i++) {
        //     std::cout <<"Iteration: " << i <<"\ttriangle1: " << cost_[i] <<"\ttriangle2: " << cost_[i+20] << std::endl; 
        // }

        double d_new[6];
        hipMemcpy(d_new, d_d, 6*sizeof(double), hipMemcpyDeviceToHost);
        for(int i=0;i<2;i++) {
            std::cout << d_new[i*3] << " " << d_new[i*3+1] << " " << d_new[i*3+2] << std::endl;
        }
        
        std::cout << "\n";


        double g[12];
        hipMemcpy(g, d_determinant, 12*sizeof(double), hipMemcpyDeviceToHost);
        for(int i=0;i<12;i++) {
            std::cout << g[i] << std::endl;
        }
        
        std::cout << "\n";

        // double gg[12];
        // hipMemcpy(gg, d_c_unit, number_triangles_*3*sizeof(double), hipMemcpyDeviceToHost);
        // for(int i=0;i<2;i++) {
        //     std::cout << gg[i*3] << " " << gg[i*3+1] << " " << gg[i*3+2] << " " << std::endl;
        // }
std::cout << "\n";std::cout << "\n";std::cout << "\n";
        double hh[18];
        hipMemcpy(hh, d_H, 18*sizeof(double), hipMemcpyDeviceToHost);
        for(int i=0;i<6;i++) {
            std::cout << hh[i*3] << " " << hh[i*3+1] << " " << hh[i*3+2] << " " << std::endl;
        }
        
        std::cout << "\n";


        double hhh[16];
        hipMemcpy(hhh, d_AWA, 16*sizeof(double), hipMemcpyDeviceToHost);
        for(int i=0;i<4;i++) {
            std::cout << hhh[i*4] << "\t" << hhh[i*4+1] << "\t" << hhh[i*4+2] << "\t" << hhh[i*4+3] << "\t" << std::endl;
        }
        
        std::cout << "\n";

        double hhhh[4];
        hipMemcpy(hhhh, d_AWb, 4*sizeof(double), hipMemcpyDeviceToHost);
        for(int i=0;i<4;i++) {
            std::cout << hhhh[i] << std::endl;
        }
        
        std::cout << "\n";



        // double *h_vertices;
        // h_vertices = (double*)malloc(number_vertices_*3*sizeof(double));
        // hipMemcpy(h_vertices, d_vertices_, number_vertices_*3*sizeof(double), hipMemcpyDeviceToHost);

        // for(int i=0; i<number_vertices_;i++) {
        //     std::cout << h_vertices[i*3+0] << " " << h_vertices[i*3+1] << " " << h_vertices[i*3+2] << std::endl;
        // }
        // exit(1);

        // double in[9];

        // hipMemcpy(in, d_information_matrix, 2*9*sizeof(double), hipMemcpyDeviceToHost);
        // for (int i = 0; i< 9*2; i++) {

        //     if(i==9) {
        //         std::cout << std::endl;
        //     }
        //     std::cout << in[i] << std::endl;
        // }
        // std::cout << "\n";

        // double in1[4];
        // hipMemcpy(in, d_b, 4*sizeof(double), hipMemcpyDeviceToHost);
        // for(int i=0;i<4;i++) {
        //     std::cout << in1[i] << std::endl;
        // }
        // for(int i=0;i<number_triangles_;i++) {
        //     for(int m=0;m<3;m++) {
        //         for(int n=0;n<3;n++) {
        //             std::cout << in[i*9+m*3+n] << "\t";
        //         }
        //         std::cout << std::endl;
        //     }
        //         std::cout << std::endl;

        // }



        // double test[4];
        // hipMemcpy(test, d_b, 4 * sizeof(double), hipMemcpyDeviceToHost);

        // for(int i=0;i<4;i++) {
        //     std::cout << test[i] << std::endl;
        // }
        
        // double* vertices_ = (double*)malloc(number_vertices_*3*sizeof(double)); // ist ok

        // hipMemcpy(vertices_, d_obs_,  number_vertices_*2*sizeof(double), hipMemcpyDeviceToHost);
        // for(int i=0;i<number_vertices_;i++) {
        //     std::cout << vertices_[i*3] << " " << vertices_[i*3+1] << " " << vertices_[i*3+1] << std::endl;
        // }
    }



}

__device__ void existIDInW(int tri1, int tri2, int tri3, int val, bool &isIn, int &tri) {
    isIn = false;

    if(tri1 == val) {
        isIn = true;
        tri = 0;
    } else if(tri2 == val) {
        isIn = true;
        tri = 1;
    } else if(tri3 == val) {
        isIn = true;
        tri = 2;
    }
}

__global__ void prepareCuSolver(int* num_triangles, int* m, int* triangles, int* colPtrCsr, int* rowPtrCsr, double* valPtrCsr, double* b, double* info_matrix, double* d, int* nnz) {

    // m = num_vertices!

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < *nnz) {
        prepare_AWA(num_triangles, triangles, info_matrix, m, colPtrCsr, rowPtrCsr, valPtrCsr);
    } else if (tid < (*nnz + *m)) {
        prepare_AWb(triangles, num_triangles, info_matrix, d, b, nnz);
    }
}

__device__ void prepare_AWA(int* num_triangles, int* triangles, double* info_matrix, int* m, int* colPtrCsr, int* rowPtrCsr, double* valPtrCsr){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int colIdx = colPtrCsr[tid];
    int rowIdx = -1;
    valPtrCsr[tid] = 0;
    for(int i=0; i < *m; i++) {
        if((tid >= rowPtrCsr[i]) && (tid < rowPtrCsr[i+1])) {
            rowIdx = i;
            // colIdx = colPtrCsr[tid];
            break;
        }
    }

    for(int i=0; i < *num_triangles; i++) {
        int tri1, tri2, tri3;
        bool rowExist = false;
        bool colExist = false;

        int rIdx = -1;
        int cIdx = -1;

        tri1 = triangles[i*3];
        tri2 = triangles[i*3+1];
        tri3 = triangles[i*3+2];
        
        existIDInW(tri1, tri2, tri3, rowIdx, rowExist, rIdx);
        existIDInW(tri1, tri2, tri3, colIdx, colExist, cIdx);

        if (rowExist && colExist) {
            
            double* info_mat_tmp = info_matrix + (i*9);
            valPtrCsr[tid] += info_mat_tmp[rIdx*3+cIdx]; // row or col IDx kann größer als drei sein!
        }
    }
}


__device__ void prepare_AWb(int* triangles, int* num_triangles, double* info_matrix, double* d, double* b, int* nnz)  {
    int tid = blockDim.x * blockIdx.x + threadIdx.x - *nnz;
    b[tid] = 0;

    for(int i=0; i < *num_triangles; i++) {
        int tri1, tri2, tri3;
        double* info_mat_tmp;
        double* d_tmp;
        tri1 = triangles[i*3];
        tri2 = triangles[i*3+1];
        tri3 = triangles[i*3+2];

        if(tid==tri1) {
            info_mat_tmp = info_matrix + (i*9);
            d_tmp = d + (i*3);
            b[tid] +=  info_mat_tmp[0] * d_tmp[0];
            b[tid] +=  info_mat_tmp[3] * d_tmp[1];
            b[tid] +=  info_mat_tmp[6] * d_tmp[2];
        } else if(tid==tri2) {
            info_mat_tmp = info_matrix + (i*9);
            d_tmp = d + (i*3);
            b[tid] += info_mat_tmp[1] * d_tmp[0];
            b[tid] += info_mat_tmp[4] * d_tmp[1];
            b[tid] += info_mat_tmp[7] * d_tmp[2];
        } else if(tid==tri3) {
            info_mat_tmp = info_matrix + (i*9);
            d_tmp = d + (i*3);
            b[tid] += info_mat_tmp[2] * d_tmp[0];
            b[tid] += info_mat_tmp[5] * d_tmp[1];
            b[tid] += info_mat_tmp[8] * d_tmp[2];
        }
        
    }
}


// __global__ void obs2unitvector(double* obs, double* vertices, double* K, int* number_obs) {
    
//     int tid = blockDim.x * blockIdx.x + threadIdx.x;
//     if(tid > *number_obs)
//         return;

//     int obs_id = tid;

//     double uvt[3];
//     uvt[0] = (obs[obs_id*2] - K[1])/K[0];
//     uvt[1] = (obs[obs_id*2+1] - K[3])/K[2];

//     double d = sqrt(uvt[0]*uvt[0]+uvt[1]*uvt[1]+1); // get the distance to compute the unit vector!

//     uvt[0] /= d;
//     uvt[1] /= d;
//     uvt[2] = 1 / d;

//     vertices[obs_id*3]   = uvt[0];
//     vertices[obs_id*3+1] = uvt[1];
//     vertices[obs_id*3+2] = uvt[2];
// }



/*
 * at the moment needed
 * triangles, reference, vertices, max_iterations 
 * 
 * Not needed at the moment
 * K, observation, int num_obs
 */
__global__ void gauss_newton(int* triangles, double* d_bar_, double* reference, double* vertices, double* K,
                       int* num_triangles, int* num_vertices, double* observation, int* num_obs, int* max_iteration, double* cost_output, double* info_matrix, double* final_d) {


// reinterpret_cast<double*>(sharedMem + 9 * maxTrianglePerBlock*  sizeof(double));

    // create sub triangles
    int tid = threadIdx.x % 18;
    int maxTrianglePerBlock = blockDim.x / 18; // example 17 per block 
    int triangleIdInBlock = threadIdx.x / 18; // triangle id in the actual block
    int totalTriangleId = blockIdx.x * maxTrianglePerBlock + triangleIdInBlock; // total id of the triangle

    if (totalTriangleId >= *num_triangles) {
        totalTriangleId = 0;
        tid = 255;
    }

    extern __shared__ char sharedMem[];

    double *sub_u = reinterpret_cast<double*>(sharedMem); 
    double *d_bar = sub_u + 9 * maxTrianglePerBlock; // \bar{d_{ij}}
    double *d = d_bar + 3 * maxTrianglePerBlock; // d <----- variable which will be optimised!
    double *constant_unit = d + 3 * maxTrianglePerBlock; // value of the 2 unit vector  u1^T * u2 and so on
    double *cost = constant_unit + 3 * maxTrianglePerBlock;

    double *dx = cost + 20 * maxTrianglePerBlock;
    double *g = dx + 3 * maxTrianglePerBlock;
    double *g_tmp = g + 3 * maxTrianglePerBlock;
    double *H = g_tmp + 6 * maxTrianglePerBlock;
    double *H_tmp = H + 9 * maxTrianglePerBlock;


    // allocate memory space to make the GPU programming more readable!
    // extern __shared__ char sharedMem[];
    // // double *sharedDouble2 = reinterpret_cast<double*>(sharedMem + 9 * maxTrianglePerBlock*  sizeof(double));

    // double *sub_u = reinterpret_cast<double*>(sharedMem); // is this necessary?
    // double *d_bar = reinterpret_cast<double*>(sub_u + 9 * maxTrianglePerBlock*  sizeof(double)); // \bar{d_{ij}}
    // double *d = reinterpret_cast<double*>(d_bar + 3 * maxTrianglePerBlock*  sizeof(double)); // d <----- variable which will be optimised!
    // double *constant_unit = reinterpret_cast<double*>(d + 3 * maxTrianglePerBlock*  sizeof(double)); // value of the 2 unit vector  u1^T * u2 and so on
    // double *cost = reinterpret_cast<double*>(constant_unit + 3 * maxTrianglePerBlock*  sizeof(double));

    // double *dx   = reinterpret_cast<double*>(cost + 20 * maxTrianglePerBlock*  sizeof(double));   
    // double *g     = reinterpret_cast<double*>(dx + 3 * maxTrianglePerBlock*  sizeof(double));
    // double *g_tmp = reinterpret_cast<double*>(g + 3 * maxTrianglePerBlock*  sizeof(double));
    // double *H    = reinterpret_cast<double*>(g_tmp + 6 * maxTrianglePerBlock*  sizeof(double));
    // double *H_tmp = reinterpret_cast<double*>(H + 9 * maxTrianglePerBlock*  sizeof(double));
    
    // __shared__ double sub_u[9]; // is this necessary?
    // __shared__ double d_bar[3]; // \bar{d_{ij}}
    // __shared__ double d[3]; // d <----- variable which will be optimised!
    // __shared__ double constant_unit[3]; // value of the 2 unit vector  u1^T * u2 and so on
    // __shared__ double cost[20];

    // __shared__ double dx[3];   

    // __shared__ double g[3];
    // __shared__ double g_tmp[6];

    // __shared__ double H[9];
    // __shared__ double H_tmp[9];
    // cost_output[0] = 1000;

    // vertices[0]= 20;

    int max_iter = *max_iteration;


    // todo!

    // passe den zugriff auf die triangles an beispiel mit 586 triangles! ->  für 18 threads
    // allokierung ist falsch!


    int triangle[3]; 
    triangle[0] = triangles[totalTriangleId*3];
    triangle[1] = triangles[totalTriangleId*3+1];
    triangle[2] = triangles[totalTriangleId*3+2];

    // only the first thread of each block shall do that!
    

    // sub_ver[0] = vertices[triangle1*3+0];
    // sub_ver[1] = vertices[triangle1*3+1];
    // sub_ver[2] = vertices[triangle1*3+2];

    // sub_ver[3] = verticesg_tmp[triangle3*3+0];
    // sub_ver[7] = vertices[triangle3*3+1];
    // sub_ver[8] = vertices[triangle3*3+2];

    // d_bar[0] = reference[triangle1*3];
    // d_bar[1] = reference[triangle2*3];
    // d_bar[2] = reference[triangle3*3];

    

    if(tid < 3) {
        d_bar[triangleIdInBlock * 3 + tid] = d_bar_[totalTriangleId*3+tid];
    } else if(tid < 6) {
        double ref_tmp[3];
        ref_tmp[0] = reference[triangle[tid-3]*3];
        ref_tmp[1] = reference[triangle[tid-3]*3+1];
        ref_tmp[2] = reference[triangle[tid-3]*3+2];
        d[triangleIdInBlock * 3 + tid-3] = sqrt(ref_tmp[0]*ref_tmp[0] + ref_tmp[1]*ref_tmp[1] + ref_tmp[2]*ref_tmp[2]) + 1; //sqrt() -> get distance!
    } else if(tid < 9) {
        sub_u[triangleIdInBlock * 9 + tid-6] = vertices[triangle[0]*3+tid-6];
    } else if(tid < 12) {
        sub_u[triangleIdInBlock * 9 + tid-6] = vertices[triangle[1]*3+tid-9];
    } else if(tid < 15) {
        sub_u[triangleIdInBlock * 9 + tid-6] = vertices[triangle[2]*3+tid-12];
    } 
    else {
        // d[2] = 1;
        double v1[3], v2[3];
        switch (tid)
        {
        case 15:
            v1[0] = vertices[triangle[0]*3];
            v1[1] = vertices[triangle[0]*3+1];
            v1[2] = vertices[triangle[0]*3+2];

            v2[0] = vertices[triangle[1]*3];
            v2[1] = vertices[triangle[1]*3+1];
            v2[2] = vertices[triangle[1]*3+2];

            constant_unit[triangleIdInBlock * 3 + 0] = v1[0]*v2[0] + v1[1]*v2[1] + v1[2]*v2[2]; 
            break;

        case 16:
            v1[0] = vertices[triangle[0]*3];
            v1[1] = vertices[triangle[0]*3+1];
            v1[2] = vertices[triangle[0]*3+2];

            v2[0] = vertices[triangle[2]*3];
            v2[1] = vertices[triangle[2]*3+1];
            v2[2] = vertices[triangle[2]*3+2];

            constant_unit[triangleIdInBlock * 3 + 1] = v1[0]*v2[0] + v1[1]*v2[1] + v1[2]*v2[2]; 
            break;

        case 17:
            v1[0] = vertices[triangle[1]*3];
            v1[1] = vertices[triangle[1]*3+1];
            v1[2] = vertices[triangle[1]*3+2];

            v2[0] = vertices[triangle[2]*3];
            v2[1] = vertices[triangle[2]*3+1];
            v2[2] = vertices[triangle[2]*3+2];

            constant_unit[triangleIdInBlock * 3 + 2] = v1[0]*v2[0] + v1[1]*v2[1] + v1[2]*v2[2]; 
            break;
        }
    }


    // make sure sub triangles are in shared memory
    


    // optimisation
    for(int iter=0; iter < 1; iter++) {
        __syncthreads();

        // cost_output[blockIdx.x*20+iter] = 10;

        // compute cost
        // Todo: set a number of threads
        
        if(tid < 1) {
            cost[triangleIdInBlock*20+iter] = compute_cost(sub_u + triangleIdInBlock* 9, d + triangleIdInBlock* 3, d_bar + triangleIdInBlock * 3);
            // cost_output[blockIdx.x*20+iter] = vertices[2];
            cost_output[totalTriangleId*20+iter] = cost[triangleIdInBlock*20+iter];
        }
        // compute g // Todo:
        else if(tid < 7) {
            compute_g(sub_u + triangleIdInBlock*9, d + triangleIdInBlock*3, d_bar + triangleIdInBlock*3, constant_unit + triangleIdInBlock*3, g_tmp + triangleIdInBlock*6);
        }
        // compute H // Todo:
        else if(tid < 16) {
            compute_H(sub_u + triangleIdInBlock*9, d + triangleIdInBlock*3, d_bar + triangleIdInBlock*3, constant_unit + triangleIdInBlock*3, H_tmp + triangleIdInBlock*9);
        }

        __syncthreads();
        
        

        // // // bring g_tmp and H_tmp together!
        if (tid < 3) {
            merge_g(tid, g + triangleIdInBlock*3, g_tmp + triangleIdInBlock*6);

        } else if(tid < 9) {
            
            merge_H(tid, H + triangleIdInBlock*9, H_tmp + triangleIdInBlock*9);
        }

        __syncthreads();

        // if(tid < 9) {
        //     info_matrix[totalTriangleId*9+tid] = H[triangleIdInBlock * 9 + tid];
        // }
        
        // __syncthreads();
    
        if (tid < 6){
            compute_determinante(H + triangleIdInBlock*9, g_tmp + triangleIdInBlock*6);
        } else if (tid < 15) {
            compute_adjugate(H + triangleIdInBlock*9, H_tmp + triangleIdInBlock*9);
        }
        __syncthreads();
        // if(tid < 9) {
        //     info_matrix[totalTriangleId*9+tid] = H_tmp[triangleIdInBlock * 9 + tid];
        // }
        // //     double dd = (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
        // //     cost_output[tid] = H_tmp[tid];
        // // } else if(tid<10) {
        // //     cost_output[tid] = (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
        // // } 
        // // return;

        compute_update(tid, dx + triangleIdInBlock*3, d + triangleIdInBlock*3, H_tmp + triangleIdInBlock*9, g_tmp + triangleIdInBlock*6, g + triangleIdInBlock*3);

        // // switch (tid)
        // // {
        // //     case 0:
        // //         dx[0] = (H_tmp[0]*g[0] + H_tmp[3]*g[1] + H_tmp[6]*g[2]) / (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
        // //         d[0] += dx[0];
        // //         break;
        // //     case 1:
        // //         dx[1] = (H_tmp[1]*g[0] + H_tmp[4]*g[1] + H_tmp[7]*g[2]) / (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
        // //         d[1] += dx[1];
        // //         break;
        // //     case 2:
        // //         dx[2] = (H_tmp[2]*g[0] + H_tmp[5]*g[1] + H_tmp[8]*g[2]) / (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
        // //         d[2] += dx[2];
        // //         break;
            
        // //     default:
        // //         break;
        // // }


    }



    // save information matrix!
    if(tid < 9) {
        info_matrix[totalTriangleId*9+tid] = H[triangleIdInBlock * 9 + tid];
    } else if(tid < 12) {
        final_d[totalTriangleId*3+tid-9] = d[triangleIdInBlock * 3 + tid-9];
    }

}

__device__ void compute_adjugate(double* H, double* a_tmp) {
    int tid = (threadIdx.x%18)-6;

    switch (tid)
    {
    case 0:
        a_tmp[0] = H[4] * H[8] - H[7] * H[5];
        break;
    case 1:
        a_tmp[1] = H[7] * H[2] - H[1] * H[8];
        break;
    case 2:
        a_tmp[2] = H[1] * H[5] - H[4] * H[2];
        break;
    case 3:
        a_tmp[3] = H[6] * H[5] - H[3] * H[8];
        break;
    case 4:
        a_tmp[4] = H[0] * H[8] - H[2] * H[6];
        break;
    case 5:
        a_tmp[5] = H[3] * H[2] - H[0] * H[5];
        break;
    case 6:
        a_tmp[6] = H[3] * H[7] - H[6] * H[4];
        break;
    case 7:
        a_tmp[7] = H[6] * H[1] - H[0] * H[7];
        break;
    case 8:
        a_tmp[8] = H[0] * H[4] - H[3] * H[1];
        break;
    
    default:
        break;
    }
}

// computes aonly a part of the determinant
__device__ void compute_determinante(double* H, double* d_tmp) {
    int tid = threadIdx.x%18;

    switch (tid)
    {
    case 0:
        d_tmp[0] = H[0] * H[4] * H[8];
        break;
    case 1:
        d_tmp[1] = H[3] * H[7] * H[2];
        break;
    case 2:
        d_tmp[2] = H[6] * H[1] * H[5];
        break;
    case 3:
        d_tmp[3] = -H[2] * H[4] * H[6];
        break;
    case 4:
        d_tmp[4] = -H[5] * H[7] * H[0];
        break;
    case 5:
        d_tmp[5] = -H[8] * H[1] * H[3];
        break;
    
    default:
        break;
    }
}

__device__ void compute_update(int tid, double *dx, double * d, double *H_tmp, double *g_tmp, double *g) {
    switch (tid)
    {
        case 0:
            dx[0] = (H_tmp[0]*g[0] + H_tmp[3]*g[1] + H_tmp[6]*g[2]) / (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
            d[0] += dx[0];
            break;
        case 1:
            dx[1] = (H_tmp[1]*g[0] + H_tmp[4]*g[1] + H_tmp[7]*g[2]) / (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
            d[1] += dx[1];
            break;
        case 2:
            dx[2] = (H_tmp[2]*g[0] + H_tmp[5]*g[1] + H_tmp[8]*g[2]) / (g_tmp[0]+g_tmp[1]+g_tmp[2]+g_tmp[3]+g_tmp[4]+g_tmp[5]);
            d[2] += dx[2];
            break;
        
        default:
            break;
    }
}


__device__ void compute_H(double* sub_u, double* d, double* d_bar, double* constant_unit, double* H_tmp){
    double v1v2[3];
    double tmp;
    int tid = (threadIdx.x%18) - 7;

    switch (tid)
    {
    case 0:
        v1v2[0] = sub_u[0] * d[0] - sub_u[3] * d[1]; 
        v1v2[1] = sub_u[1] * d[0] - sub_u[4] * d[1];
        v1v2[2] = sub_u[2] * d[0] - sub_u[5] * d[1];

        tmp = d[0] - constant_unit[0]*d[1];
        H_tmp[0] = tmp*tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 1:
        v1v2[0] = sub_u[0] * d[0] - sub_u[3] * d[1]; 
        v1v2[1] = sub_u[1] * d[0] - sub_u[4] * d[1];
        v1v2[2] = sub_u[2] * d[0] - sub_u[5] * d[1];

        tmp = -constant_unit[0] * d[0] + d[1];
        H_tmp[1] = tmp*tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 2:
        v1v2[0] = sub_u[0] * d[0] - sub_u[3] * d[1]; 
        v1v2[1] = sub_u[1] * d[0] - sub_u[4] * d[1];
        v1v2[2] = sub_u[2] * d[0] - sub_u[5] * d[1];

        tmp = (d[0] - constant_unit[0]*d[1]) * (-constant_unit[0] * d[0] + d[1]);
        H_tmp[2] = tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 3:
        v1v2[0] = sub_u[0] * d[0] - sub_u[6] * d[2]; 
        v1v2[1] = sub_u[1] * d[0] - sub_u[7] * d[2];
        v1v2[2] = sub_u[2] * d[0] - sub_u[8] * d[2];

        tmp = d[0] - constant_unit[1]*d[2];
        H_tmp[3] = tmp*tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 4:
        v1v2[0] = sub_u[0] * d[0] - sub_u[6] * d[2]; 
        v1v2[1] = sub_u[1] * d[0] - sub_u[7] * d[2];
        v1v2[2] = sub_u[2] * d[0] - sub_u[8] * d[2];

        tmp = -constant_unit[1] * d[0] + d[2];
        H_tmp[4] = tmp*tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 5:
        v1v2[0] = sub_u[0] * d[0] - sub_u[6] * d[2]; 
        v1v2[1] = sub_u[1] * d[0] - sub_u[7] * d[2];
        v1v2[2] = sub_u[2] * d[0] - sub_u[8] * d[2];

        tmp = (d[0] - constant_unit[1]*d[2]) * (-constant_unit[1] * d[0] + d[2]);
        H_tmp[5] = tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 6:
        v1v2[0] = sub_u[3] * d[1] - sub_u[6] * d[2]; 
        v1v2[1] = sub_u[4] * d[1] - sub_u[7] * d[2];
        v1v2[2] = sub_u[5] * d[1] - sub_u[8] * d[2];

        tmp = d[1] - constant_unit[2]*d[2];
        H_tmp[6] = tmp*tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 7:
        v1v2[0] = sub_u[3] * d[1] - sub_u[6] * d[2]; 
        v1v2[1] = sub_u[4] * d[1] - sub_u[7] * d[2];
        v1v2[2] = sub_u[5] * d[1] - sub_u[8] * d[2];

        tmp = -constant_unit[2] * d[1] + d[2];
        H_tmp[7] = tmp*tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    case 8:
        v1v2[0] = sub_u[3] * d[1] - sub_u[6] * d[2]; 
        v1v2[1] = sub_u[4] * d[1] - sub_u[7] * d[2];
        v1v2[2] = sub_u[5] * d[1] - sub_u[8] * d[2];

        tmp = (d[1] - constant_unit[2]*d[2]) * (-constant_unit[2] * d[1] + d[2]);
        H_tmp[8] = tmp / (v1v2[0]*v1v2[0]+v1v2[1]*v1v2[1]+v1v2[2]*v1v2[2]);
        break;
    }
}

__device__ void merge_g(int tid, double *g, double * g_tmp) {
     switch (tid)
    {
    case 0:
        g[0] = g_tmp[0] + g_tmp[2];
        break;
    case 1:
        g[1] = g_tmp[1] + g_tmp[4];
        break;
    case 2:
        g[2] = g_tmp[3] + g_tmp[5];
        break;
    default:
        break;
    }
}

__device__ void merge_H(int tid, double *H, double * H_tmp) {
    switch (tid-3)
    {
    case 0:
        H[0] = H_tmp[0]+H_tmp[3];
        break;
    
    case 1:
        H[4] = H_tmp[1]+H_tmp[6];
        break;
    
    case 2:
        H[8] = H_tmp[4]+H_tmp[7];
        break;
    
    case 3:
        H[1] = H_tmp[2];
        H[3] = H_tmp[2];
        break;
    
    case 4:
        H[2] = H_tmp[5];
        H[6] = H_tmp[5];
        break;
    
    case 5:
        H[5] = H_tmp[8];
        H[7] = H_tmp[8];
        break;
    default:
        break;
    }
}

__device__ void compute_g(double* sub_u, double* d, double* d_bar, double* constant_unit, double* g_tmp){
    double v1v2[3];
    double normv1v2;
    
    
    int tid = (threadIdx.x % 18) - 1; // thread for each computed triangle in this block!

    switch (tid)
    {
    case 0:
        v1v2[0] = sub_u[0]*d[0]-sub_u[3]*d[1];
        v1v2[1] = sub_u[1]*d[0]-sub_u[4]*d[1];
        v1v2[2] = sub_u[2]*d[0]-sub_u[5]*d[1];
        normv1v2 = sqrt(v1v2[0]*v1v2[0] + v1v2[1]*v1v2[1] + v1v2[2]*v1v2[2]);
        g_tmp[0] = (-d[0] + constant_unit[0] * d[1])*(1 - d_bar[0] / normv1v2); 
        break;
    case 1:
        v1v2[0] = sub_u[0]*d[0]-sub_u[3]*d[1];
        v1v2[1] = sub_u[1]*d[0]-sub_u[4]*d[1];
        v1v2[2] = sub_u[2]*d[0]-sub_u[5]*d[1];
        normv1v2 = sqrt(v1v2[0]*v1v2[0] + v1v2[1]*v1v2[1] + v1v2[2]*v1v2[2]);
        g_tmp[1] = (constant_unit[0] * d[0] - d[1])*(1 - d_bar[0] / normv1v2); 
        break;
    case 2:
        v1v2[0] = sub_u[0]*d[0]-sub_u[6]*d[2];
        v1v2[1] = sub_u[1]*d[0]-sub_u[7]*d[2];
        v1v2[2] = sub_u[2]*d[0]-sub_u[8]*d[2];
        normv1v2 = sqrt(v1v2[0]*v1v2[0] + v1v2[1]*v1v2[1] + v1v2[2]*v1v2[2]);
        g_tmp[2] = (-d[0] + constant_unit[1] * d[2])*(1 - d_bar[1] / normv1v2); 
        break;
    case 3:
        v1v2[0] = sub_u[0]*d[0]-sub_u[6]*d[2];
        v1v2[1] = sub_u[1]*d[0]-sub_u[7]*d[2];
        v1v2[2] = sub_u[2]*d[0]-sub_u[8]*d[2];
        normv1v2 = sqrt(v1v2[0]*v1v2[0] + v1v2[1]*v1v2[1] + v1v2[2]*v1v2[2]);
        g_tmp[3] = (constant_unit[1] * d[0] - d[2])*(1 - d_bar[1] / normv1v2); 
        break;
    case 4:
        v1v2[0] = sub_u[3]*d[1]-sub_u[6]*d[2];
        v1v2[1] = sub_u[4]*d[1]-sub_u[7]*d[2];
        v1v2[2] = sub_u[5]*d[1]-sub_u[8]*d[2];
        normv1v2 = sqrt(v1v2[0]*v1v2[0] + v1v2[1]*v1v2[1] + v1v2[2]*v1v2[2]);
        g_tmp[4] = (-d[1] + constant_unit[2] * d[2])*(1 - d_bar[2] / normv1v2); 
        break;
    case 5:
        v1v2[0] = sub_u[3]*d[1]-sub_u[6]*d[2];
        v1v2[1] = sub_u[4]*d[1]-sub_u[7]*d[2];
        v1v2[2] = sub_u[5]*d[1]-sub_u[8]*d[2];
        normv1v2 = sqrt(v1v2[0]*v1v2[0] + v1v2[1]*v1v2[1] + v1v2[2]*v1v2[2]);
        g_tmp[5] = (constant_unit[2] * d[1] - d[2])*(1 - d_bar[2] / normv1v2); 
        break;
    }
}


__device__ double compute_cost(double* sub_u, double* d, double* d_bar) {
    double error[3];
    double v1[3], v2[3], v3[3];

    v1[0]= sub_u[0]*d[0];
    v1[1]= sub_u[1]*d[0];
    v1[2]= sub_u[2]*d[0];

    v2[0]= sub_u[3]*d[1];
    v2[1]= sub_u[4]*d[1];
    v2[2]= sub_u[5]*d[1];

    v3[0]= sub_u[6]*d[2];
    v3[1]= sub_u[7]*d[2];
    v3[2]= sub_u[8]*d[2];

    error[0] = sqrt((v1[0]-v2[0])*(v1[0]-v2[0])+(v1[1]-v2[1])*(v1[1]-v2[1])+(v1[2]-v2[2])*(v1[2]-v2[2])) - d_bar[0];
    error[1] = sqrt((v1[0]-v3[0])*(v1[0]-v3[0])+(v1[1]-v3[1])*(v1[1]-v3[1])+(v1[2]-v3[2])*(v1[2]-v3[2])) - d_bar[1];
    error[2] = sqrt((v2[0]-v3[0])*(v2[0]-v3[0])+(v2[1]-v3[1])*(v2[1]-v3[1])+(v2[2]-v3[2])*(v2[2]-v3[2])) - d_bar[2];

    return sqrt(error[0]*error[0]+error[1]*error[1]+error[2]*error[2]);
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void compute_AWB(double *info_matrix, double *d, int *triangle, int *num_triangle, int *numVertices, double *AWB) {
    int global_thread = blockDim.x * blockIdx.x + threadIdx.x;

    // int triangle_id = global_thread / (*num_triangle * 3);

    // if(triangle_id >= *num_triangle)
    //     return;

    // int global_thread = blockDim.x * blockIdx.x + threadIdx.x;

    int triangle_id = global_thread / 3;

    if (triangle_id >= *num_triangle)
        return;

    int tid1 = global_thread % 3;
    int tid2 = (tid1 + 1) % 3;
    int tid3 = (tid1 + 2) % 3;

    // for(int i=0; i < *num_triangle; i++) {

        int tri1 = triangle[triangle_id * 3 + tid1];    
        int tri2 = triangle[triangle_id * 3 + tid2];    
        int tri3 = triangle[triangle_id * 3 + tid3];

        double tid1_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid1];
        double tid2_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid2];
        double tid3_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid3];


        double d1 = d[triangle_id * 3 + tid1];
        double d2 = d[triangle_id * 3 + tid2];
        double d3 = d[triangle_id * 3 + tid3];

        atomicAdd_double(&AWB[tri1], (tid1_val * d1) + (tid2_val * d2) + (tid3_val * d3));

    // }


    // int tid1 = global_thread % 3;
    // int tid2 = (tid1 + 1) % 3;
    // int tid3 = (tid1 + 2) % 3;

    // for(int i=0; i < *num_triangle; i++) {

    //     int tri1 = triangle[i * 3 + tid1];    
    //     int tri2 = triangle[i * 3 + tid2];    
    //     int tri3 = triangle[i * 3 + tid3];

    //     double tid1_val = info_matrix[i * 9 + tid1 * 3 + tid1];
    //     double tid2_val = info_matrix[i * 9 + tid1 * 3 + tid2];
    //     double tid3_val = info_matrix[i * 9 + tid1 * 3 + tid3];


    //     double d1 = d[i * 3 + tid1];
    //     double d2 = d[i * 3 + tid2];
    //     double d3 = d[i * 3 + tid3];

    //     AWB[tri1] +=    (tid1_val * d1) + 
    //                     (tid2_val * d2) + 
    //                     (tid3_val * d3);

    // }

}

__device__ double atomicAdd_double(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void compute_AWA(double *info_matrix, int *triangle, int *num_triangle, int *numVertices, double *AWA) {

    // int global_thread = blockDim.x * blockIdx.x + threadIdx.x;

    // int triangle_id = global_thread / (*num_triangle * 3);

    // if(triangle_id >= *num_triangle)
    //     return;


    int global_thread = blockDim.x * blockIdx.x + threadIdx.x;

    int triangle_id = global_thread / 3;

    if (triangle_id >= *num_triangle)
        return;

    int tid1 = global_thread % 3;
    int tid2 = (tid1 + 1) % 3;
    int tid3 = (tid1 + 2) % 3;

    int tri1 = triangle[triangle_id * 3 + tid1];    
    int tri2 = triangle[triangle_id * 3 + tid2];    
    int tri3 = triangle[triangle_id * 3 + tid3];

    double tid1_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid1];
    double tid2_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid2];
    double tid3_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid3];


        atomicAdd_double(&AWA[tri1 * *numVertices + tri1], tid1_val);
        atomicAdd_double(&AWA[tri1 * *numVertices + tri2], tid2_val);
        // AWA[tri1 * *numVertices + tri3] += tid3_val;
        atomicAdd_double(&AWA[tri2 * *numVertices + tri1], tid2_val);
        // AWA[tri3 * *numVertices + tri1] += tid3_val;

    // int tid1 = global_thread % 3;
    // int tid2 = (tid1 + 1) % 3;
    // int tid3 = (tid1 + 2) % 3;

    // for(int i=0; i < *num_triangle; i++) {

    //     int tri1 = triangle[i * 3 + tid1];    
    //     int tri2 = triangle[i * 3 + tid2];    
    //     int tri3 = triangle[i * 3 + tid3];

    //     double tid1_val = info_matrix[i * 9 + tid1 * 3 + tid1];
    //     double tid2_val = info_matrix[i * 9 + tid1 * 3 + tid2];
    //     double tid3_val = info_matrix[i * 9 + tid1 * 3 + tid3];

    //     AWA[tri1 * *numVertices + tri1] += tid1_val;
    //     AWA[tri1 * *numVertices + tri2] += tid2_val;
    //     // AWA[tri1 * *numVertices + tri3] += tid3_val;
    //     AWA[tri2 * *numVertices + tri1] += tid2_val;
    //     // AWA[tri3 * *numVertices + tri1] += tid3_val;

    // }
    

//     int tri1 = triangle[triangle_id * 3 + tid1];    
//     int tri2 = triangle[triangle_id * 3 + tid2];    
//     int tri3 = triangle[triangle_id * 3 + tid3];

//     AWA[tri1 * 4 + tri1] = 0;

//     int numVert = *numVertices;

//     double tid1_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid1];
//     double tid2_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid2];
//     double tid3_val = info_matrix[triangle_id * 9 + tid1 * 3 + tid3];

//     // info_matrix[9 + tid1 * 3 + tid1] = (double)tid1_val;
// // AWA[tri1 * 4 + tri1] = 0;
//     atomicAdd(&AWA[tri1 * numVert + tri1], tid1_val);
    
    // atomicAdd(&AWA[tri1 * numVert + tri2], tid2_val);
    // atomicAdd(&AWA[tri1 * numVert + tri3], tid3_val);
    // atomicAdd(&AWA[tri2 * numVert + tri1], tid2_val);
    // atomicAdd(&AWA[tri3 * numVert + tri1], tid3_val);
     
}

__global__ void compute_cost_new(double *d_bar, int *triangle, int *num_triangle, double *d, double *vertices, double *cost) {
    int global_thread = blockDim.x * blockIdx.x + threadIdx.x;

    int triangle_id = global_thread;

    if(triangle_id >= *num_triangle) 
        return;

       

    int tri1 = triangle[triangle_id * 3 + 0];
    int tri2 = triangle[triangle_id * 3 + 1];
    int tri3 = triangle[triangle_id * 3 + 2];

    
    double dbar1 = d_bar[triangle_id * 3 + 0];
    double dbar2 = d_bar[triangle_id * 3 + 1];
    double dbar3 = d_bar[triangle_id * 3 + 2];

    double d1 = d[triangle_id*3 + 0];
    double d2 = d[triangle_id*3 + 1];
    double d3 = d[triangle_id*3 + 2];

    double *u1 = vertices + tri1 * 3; 
    double *u2 = vertices + tri2 * 3; 
    double *u3 = vertices + tri3 * 3; 


    double v12x = u1[0] * d1 - u2[0] * d2;
    double v12y = u1[1] * d1 - u2[1] * d2;
    double v12z = u1[2] * d1 - u2[2] * d2;

    double v23x = u2[0] * d2 - u3[0] * d3;
    double v23y = u2[1] * d2 - u3[1] * d3;
    double v23z = u2[2] * d2 - u3[2] * d3;

    double v31x = u3[0] * d3 - u1[0] * d1;
    double v31y = u3[1] * d3 - u1[1] * d1;
    double v31z = u3[2] * d3 - u1[2] * d1;
    

    double e12 = dbar1 - sqrt(v12x*v12x + v12y*v12y + v12z*v12z);
    double e23 = dbar2 - sqrt(v23x*v23x + v23y*v23y + v23z*v23z);
    double e31 = dbar3 - sqrt(v31x*v31x + v31y*v31y + v31z*v31z);

    cost[triangle_id * 20 + 0] = sqrt(e12 * e12 + e23 * e23 + e31 * e31);
    

    

}

__global__ void compute_d(double *d, double *ref, int *num_triangle, int *triangles) {

    int total = blockDim.x * blockIdx.x + threadIdx.x;

    int triangle_id = total / 3;
    int tid = total % 3;

    if (triangle_id >= *num_triangle)
        return;

    
    int tri = triangles[triangle_id * 3 + tid];

    double *vertex = ref + tri * 3; 

    d[triangle_id * 3 + tid]  = sqrt(vertex[0] * vertex[0] + vertex[1] * vertex[1] + vertex[2] * vertex[2]) + 1;

}

__global__ void update_dx(double *d, int *num_triangles, double *adjugate_matrix, double *determinante, double *g, double *cost) {
    int global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;

    int tid = global_thread_id % 3;
    int triangle_id = global_thread_id / 3;

    if(triangle_id >= *num_triangles) 
        return;

    double det = determinante[triangle_id * 6];

    double dx = ((adjugate_matrix[triangle_id * 9 + tid * 3 + 0] * g[triangle_id * 3] ) +
                 (adjugate_matrix[triangle_id * 9 + tid * 3 + 1] * g[triangle_id * 3 + 1] ) +
                 (adjugate_matrix[triangle_id * 9 + tid * 3 + 2] * g[triangle_id * 3 + 2] )) / det;

    d[triangle_id*3 + tid] += dx;
    cost[triangle_id * 20 + tid + 1] = dx;

}

__global__ void compute_adjugate_new(double *H, int *num_triangles, double *adjugate_matrix) {
    int global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;

    int tid = global_thread_id % 9;
    int triangle_id = global_thread_id / 9;

    if (triangle_id >= *num_triangles)
        return;

    

    int row = tid / 3;
    int col = tid % 3;

    int m = col % 2;
    int m1 = row % 2;
    int m2 = -1+2*((tid+1)%2);

    int left = (col + 1 + 1*m) % 3;
    int right = (col + 2 + 2*m) % 3;

    int top = ((row+1+1*m1)*3) % 9;
    int bot = ((row+2+2*m1)*3) % 9;

    int a1 = top + left;
    int a2 = bot + left;
    
    int b1 = top + right;
    int b2 = bot + right;

    int start_address = triangle_id * 9;
    // adjugate_matrix[triangle_id*9+tid] = a1;

    adjugate_matrix[triangle_id*9 + col * 3 + row] = (H[start_address +a1]*H[start_address + b2] - H[start_address + b1] * H[start_address + a2]) * m2;
}


__global__ void compute_determinante_new(double *H, int *num_triangles, double *determinante) {
    int global_thread_id = blockDim.x*blockIdx.x+threadIdx.x;

    int triangle_id = global_thread_id / 6;
    int tid;
    if(triangle_id < *num_triangles) {
        tid = global_thread_id % 6;

        int m1 = tid / 3;

        int first = ((tid + 1*m1) + (tid + 1*m1)*m1) % 3;
        int second = 3 + ((tid + 1 + 1*m1) + (tid + 1 + 1*m1)*m1)%3;
        int third  = 6 + ((tid + 2 + 1*m1) + (tid + 2 + 1*m1)*m1)%3;

        determinante[triangle_id * 6 + tid] = H[triangle_id * 9 + first] * H[triangle_id * 9 + second] * H[triangle_id * 9 + third];
        // determinante[triangle_id * 6 + tid] = third;
    }

    __syncthreads();

    if ((triangle_id >= *num_triangles) || (tid >=1))
        return;

    // determinante[triangle_id * 6] = triangle_id;
    determinante[triangle_id * 6] = determinante[triangle_id * 6 + 0] + determinante[triangle_id * 6 + 1] + determinante[triangle_id * 6 + 2] - 
                                    determinante[triangle_id * 6 + 3] - determinante[triangle_id * 6 + 4] - determinante[triangle_id * 6 + 5];

}



__global__ void compute_constantUnits(double *c_unit, int *triangles, int *num_triangles, double *vertices) {

    int total_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    int triangle_id = total_thread_id / 3;
    if (triangle_id >= *num_triangles)
        return;

    int tid = total_thread_id % 3;
    int tid_next_id = (tid + 1) % 3;

    int triangle_face_id1 = triangles[triangle_id*3 + tid];
    int triangle_face_id2 = triangles[triangle_id*3 + tid_next_id]; 

    double *u1 = vertices + triangle_face_id1*3;
    double *u2 = vertices + triangle_face_id2*3;

    double value = u1[0] * u2[0] + u1[1] * u2[1] + u1[2] * u2[2];

    c_unit[triangle_id*3 + tid] = value;
}

__global__ void obs2unitvector(double* obs, double* vertices, double* K, int* number_obs) {
    
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid > *number_obs)
        return;

    int obs_id = tid;

    double uvt[3];
    uvt[0] = (obs[obs_id*2] - K[1])/K[0];
    uvt[1] = (obs[obs_id*2+1] - K[3])/K[2];

    double d = sqrt(uvt[0]*uvt[0]+uvt[1]*uvt[1]+1); // get the distance to compute the unit vector!

    uvt[0] /= d;
    uvt[1] /= d;
    uvt[2] = 1 / d;

    vertices[obs_id*3]   = uvt[0];
    vertices[obs_id*3+1] = uvt[1];
    vertices[obs_id*3+2] = uvt[2];
}


__global__ void computeH_new(double *H, int *triangles, double* vertices, double *d, double *constant_unit, int *num_triangles) {

    int total_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    int triangle_id = total_thread_id / 3;

    double J2J2;
    int tid_next_id ;
    if (triangle_id < *num_triangles) {
        

        int tid = total_thread_id % 3;
        tid_next_id = (tid + 1) % 3;

        int triangle_face_id1 = triangles[triangle_id*3 + tid];
        int triangle_face_id2 = triangles[triangle_id*3 + tid_next_id]; 

        double d1 = d[triangle_id * 3 + tid];
        double d2 = d[triangle_id * 3 + tid_next_id];

        double c_unit = constant_unit[triangle_id*3 + tid];

        double J1_part = d1 - c_unit * d2;
        double J2_part = d2 - c_unit * d1;

        double x = (vertices[triangle_face_id1*3+0] * d1 - vertices[triangle_face_id2*3+0] * d2);
        double y = (vertices[triangle_face_id1*3+1] * d1 - vertices[triangle_face_id2*3+1] * d2);
        double z = (vertices[triangle_face_id1*3+2] * d1 - vertices[triangle_face_id2*3+2] * d2);
        

        double dnorm =  sqrt(x*x + y*y + z*z);

        double J1 = J1_part/ dnorm;
        double J2 = J2_part/ dnorm;
        double J1J2 = J1 * J2;
        J2J2 = J2*J2;
        
        // safe in H matrix on diagonal
        H[triangle_id*9 + tid*3+tid] = J1*J1;
        // safe on non diagonal
        H[triangle_id*9 + tid*3+tid_next_id] = J1J2;   
        // safe on a placeholder field
        H[triangle_id*9 + tid_next_id*3+tid] = J1J2;   
    } 

    __syncthreads();

    if(triangle_id >=  *num_triangles) 
        return;

    H[triangle_id*9 + tid_next_id*3+tid_next_id] += J2J2;
    

}


__global__ void computeG_new(double *g, int *triangles, double * vertices, double *d, double *d_bar, double *constant_unit, int *num_triangles) {
    int total_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    int triangle_id = total_thread_id / 3;
    double g2;
    int tid_next_id;

    if(triangle_id <  *num_triangles){

        int tid = total_thread_id % 3;
        tid_next_id = (tid + 1) % 3;

        int triangle_face_id1 = triangles[triangle_id*3 + tid];
        int triangle_face_id2 = triangles[triangle_id*3 + tid_next_id]; 

        double d1 = d[triangle_id * 3 + tid];
        double d2 = d[triangle_id * 3 + tid_next_id];

        double c_unit = constant_unit[triangle_id*3 + tid];

        double d_bar12 = d_bar[triangle_id*3+tid];

        double x = (vertices[triangle_face_id1*3+0] * d1 - vertices[triangle_face_id2*3+0] * d2);
        double y = (vertices[triangle_face_id1*3+1] * d1 - vertices[triangle_face_id2*3+1] * d2);
        double z = (vertices[triangle_face_id1*3+2] * d1 - vertices[triangle_face_id2*3+2] * d2);
        
        double dnorm =  sqrt(x*x + y*y + z*z);

        double left_part = d_bar12 / dnorm - 1 ;

        double g1= left_part*(d1-c_unit*d2);
               g2= left_part*(d2-c_unit*d1);

        g[triangle_id * 3 + tid] = g1;
        // g[triangle_id * 3 + tid] = 0;
    }
    __syncthreads();
    
    if(triangle_id >=  *num_triangles) 
        return;

    g[triangle_id * 3 + tid_next_id] += g2;
}


__global__ void compute_d_bar_x(int* numTriangles, int *triangles, double *reference, double *d_bar) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid >=  *numTriangles)
        return;

    int triangleId = tid;
    int triangle[3];
    triangle[0] = triangles[triangleId*3];
    triangle[1] = triangles[triangleId*3+1];
    triangle[2] = triangles[triangleId*3+2];

    double v1[3];
    double v2[3];
    double tmp[3];

    v1[0] = reference[triangle[0]*3];
    v1[1] = reference[triangle[0]*3+1];
    v1[2] = reference[triangle[0]*3+2];

    v2[0] = reference[triangle[1]*3];
    v2[1] = reference[triangle[1]*3+1];
    v2[2] = reference[triangle[1]*3+2];
    
    tmp[0] = v1[0] - v2[0];
    tmp[1] = v1[1] - v2[1];
    tmp[2] = v1[2] - v2[2];
    d_bar[triangleId*3] = sqrt(tmp[0]*tmp[0] + tmp[1]*tmp[1] + tmp[2]*tmp[2]); 

}

__global__ void compute_d_bar_y(int* numTriangles, int *triangles, double *reference, double *d_bar) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid >=  *numTriangles)
        return;

    int triangleId = tid;
    int triangle[3];
    triangle[0] = triangles[triangleId*3];
    triangle[1] = triangles[triangleId*3+1];
    triangle[2] = triangles[triangleId*3+2];

    double v1[3];
    double v2[3];
    double tmp[3];

    v1[0] = reference[triangle[1]*3];
    v1[1] = reference[triangle[1]*3+1];
    v1[2] = reference[triangle[1]*3+2];

    v2[0] = reference[triangle[2]*3];
    v2[1] = reference[triangle[2]*3+1];
    v2[2] = reference[triangle[2]*3+2];
    
    tmp[0] = v1[0] - v2[0];
    tmp[1] = v1[1] - v2[1];
    tmp[2] = v1[2] - v2[2];
    d_bar[triangleId*3+1] = sqrt(tmp[0]*tmp[0] + tmp[1]*tmp[1] + tmp[2]*tmp[2]); 

}

__global__ void compute_d_bar_z(int* numTriangles, int *triangles, double *reference, double *d_bar) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid >=  *numTriangles)
        return;

    int triangleId = tid;
    int triangle[3];
    triangle[0] = triangles[triangleId*3];
    triangle[1] = triangles[triangleId*3+1];
    triangle[2] = triangles[triangleId*3+2];

    double v1[3];
    double v2[3];
    double tmp[3];

    v1[0] = reference[triangle[2]*3];
    v1[1] = reference[triangle[2]*3+1];
    v1[2] = reference[triangle[2]*3+2];

    v2[0] = reference[triangle[0]*3];
    v2[1] = reference[triangle[0]*3+1];
    v2[2] = reference[triangle[0]*3+2];
    
    tmp[0] = v1[0] - v2[0];
    tmp[1] = v1[1] - v2[1];
    tmp[2] = v1[2] - v2[2];
    d_bar[triangleId*3+2] = sqrt(tmp[0]*tmp[0] + tmp[1]*tmp[1] + tmp[2]*tmp[2]); 

}



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void compute_d_bar(int* triangles, double* reference, double* d_bar) {
    int triangle[3]; 
    triangle[0] = triangles[blockIdx.x*3];
    triangle[1] = triangles[blockIdx.x*3+1];
    triangle[2] = triangles[blockIdx.x*3+2];
    double v1[3];
    double v2[3];
    double tmp[3];

    // Error in here!
    // Todo: use fixed size arrays! so klappt das nicht! so wirft das nur ein cuda error und dann habe ich das problem!

    int tid = threadIdx.x;
    switch (tid)
    {
    case 0: // norm(v1-v2)
        v1[0] = reference[triangle[0]*3];
        v1[1] = reference[triangle[0]*3+1];
        v1[2] = reference[triangle[0]*3+2];

        v2[0] = reference[triangle[1]*3];
        v2[1] = reference[triangle[1]*3+1];
        v2[2] = reference[triangle[1]*3+2];
        
        tmp[0] = v1[0] - v2[0];
        tmp[1] = v1[1] - v2[1];
        tmp[2] = v1[2] - v2[2];
        d_bar[blockIdx.x*3] = sqrt(tmp[0]*tmp[0] + tmp[1]*tmp[1] + tmp[2]*tmp[2]); 
        break;
    case 1: // norm(v1-v3)
        v1[0] = reference[triangle[0]*3];
        v1[1] = reference[triangle[0]*3+1];
        v1[2] = reference[triangle[0]*3+2];

        v2[0] = reference[triangle[2]*3];
        v2[1] = reference[triangle[2]*3+1];
        v2[2] = reference[triangle[2]*3+2];
        
        tmp[0] = v1[0] - v2[0];
        tmp[1] = v1[1] - v2[1];
        tmp[2] = v1[2] - v2[2];
        d_bar[blockIdx.x*3+1] = sqrt(tmp[0]*tmp[0]+tmp[1]*tmp[1]+tmp[2]*tmp[2]);
        break;

    case 2: // norm(v2-v3)
        v1[0] = reference[triangle[1]*3];
        v1[1] = reference[triangle[1]*3+1];
        v1[2] = reference[triangle[1]*3+2];

        v2[0] = reference[triangle[2]*3];
        v2[1] = reference[triangle[2]*3+1];
        v2[2] = reference[triangle[2]*3+2];
        
        tmp[0] = v1[0] - v2[0];
        tmp[1] = v1[1] - v2[1];
        tmp[2] = v1[2] - v2[2];
        d_bar[blockIdx.x*3+2] = sqrt(tmp[0]*tmp[0]+tmp[1]*tmp[1]+tmp[2]*tmp[2]);
        break;
    }

}

// __global__ void tester(double* in1, double* in2,double* out) {
//     out[threadIdx.x] = in1[threadIdx.x] + in2[threadIdx.y];
// }

void checkCusolverStatus(hipsolverStatus_t status) {
    switch (status) {
        case HIPSOLVER_STATUS_SUCCESS:
            std::cout << "Operation completed successfully." << std::endl;
            break;
        case HIPSOLVER_STATUS_NOT_INITIALIZED:
            std::cout << "HIPSOLVER_STATUS_NOT_INITIALIZED: The library was not initialized." << std::endl;
            break;
        case HIPSOLVER_STATUS_ALLOC_FAILED:
            std::cout << "HIPSOLVER_STATUS_ALLOC_FAILED: The resources could not be allocated." << std::endl;
            break;
        case HIPSOLVER_STATUS_INVALID_VALUE:
            std::cout << "HIPSOLVER_STATUS_INVALID_VALUE: Invalid parameters were passed." << std::endl;
            break;
        case HIPSOLVER_STATUS_ARCH_MISMATCH:
            std::cout << "HIPSOLVER_STATUS_ARCH_MISMATCH: The device only supports compute capability 5.0 and above." << std::endl;
            break;
        case HIPSOLVER_STATUS_INTERNAL_ERROR:
            std::cout << "HIPSOLVER_STATUS_INTERNAL_ERROR: An internal operation failed." << std::endl;
            break;
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            std::cout << "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: The matrix type is not supported." << std::endl;
            break;
        default:
            std::cout << "Unknown error occurred." << std::endl;
    }
}


__global__ void addKernel(float* d_data, double increment) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
         // Ensure that only one thread modifies the value
    atomicAdd(d_data, increment);
    
}

int main() {

//  float h_data = 12.0f;  // Initial Wert auf dem Host
//     float* d_data;
//     double increment = 1.5f;  // Der Wert, den wir addieren wollen

//     // Speicher auf der GPU allozieren
//     hipMalloc(&d_data, sizeof(float));
//     hipMemcpy(d_data, &h_data, sizeof(float), hipMemcpyHostToDevice);

//     // Kernel starten
//     addKernel<<<1, 2>>>(d_data, increment);
    
//     // Warten auf Kernel-Finish
//     hipDeviceSynchronize();

//     // Daten zurück auf den Host kopieren
//     hipMemcpy(&h_data, d_data, sizeof(float), hipMemcpyDeviceToHost);

//     std::cout << "Result: " << h_data << std::endl;

//     // GPU-Speicher freigeben
//     hipFree(d_data);

// double test[1] = {123.132123};

//     // Konvertierung von double zu long long (64-Bit)
//     long long temp;
//     std::memcpy(&temp, test, sizeof(double));

//     // Konvertierung zurück zu double
//     double result;
//     std::memcpy(&result, &temp, sizeof(double));

//     std::cout << "Original Value: " << test[0] << std::endl;
//     std::cout << "Recovered Value: " << result << std::endl;
// exit(1);

    hipDeviceProp_t prop;
hipGetDeviceProperties(&prop, 0);

printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
printf("Max Threads per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
printf("Number of Multiprocessors: %d\n", prop.multiProcessorCount);
printf("Max blocks per Multiprozessor: %d\n", prop.maxBlocksPerMultiProcessor); 
printf("Warp size: %d\n", prop.warpSize);
printf("MaxThreadDim: %d\n", prop.maxThreadsDim);
// exit(1);

    std::cout << "optim start" << std::endl;
    
    std::vector<Eigen::Vector3i> triangles;
    std::vector<Eigen::Vector3d> vertices;
    

    // //  Datei öffnen
    // std::ifstream file("/home/anonym/Schreibtisch/PhD/code/GPU_Distance_only/data/phi_SfT/real/S1/templates/template_mesh_final.obj");
    
    // if (!file.is_open()) {
    //     std::cerr << "Konnte die Datei nicht öffnen!" << std::endl;
    //     return 1;
    // }

    // std::string line;
    // while (std::getline(file, line)) {
    //     // Stream zur Zeilenverarbeitung
    //     std::istringstream iss(line);
        
    //     std::string prefix;
    //     double x, y, z;  // Verwendung von double für höhere Genauigkeit
        
    //     // Erstes Element (z.B. "v") einlesen
    //     iss >> prefix;
        
    //     // Die nächsten drei Elemente (Koordinaten) einlesen
    //     if (prefix == "v") {
    //         iss >> x >> y >> z;
    //         Eigen::Vector3d v_tmp;
    //         v_tmp << x,y,z;
    //         vertices.push_back(v_tmp);
    //     }
    //     if (prefix == "f"){
    //         std::string block;
    //         int f1, f2, f3;
    //         int counter = 0;
    //         while (iss >> block) {  // Blöcke einzeln lesen
    //             std::istringstream blockStream(block);
    //             std::string firstNumber;
                
    //             // Erste Zahl extrahieren (vor dem '/')
    //             std::getline(blockStream, firstNumber, '/');

    //             // std::cout << std::stoi(firstNumber) << std::endl;
                
    //             switch (counter)
    //             {
    //             case 0:
    //                 f1 = std::stoi(firstNumber);
    //                 break;
    //             case 1:
    //                 f2 = std::stoi(firstNumber);
    //                 break;
    //             case 2:
    //                 f3 = std::stoi(firstNumber);
    //                 break;
                
    //             default:
    //                 break;
    //             }
    //             counter++;
                
                
    //         }
    //         // exit(1);
    //         // iss >> x >> y >> z;
    //         Eigen::Vector3i t_tmp;
    //         t_tmp << f1-1,f2-1,f3-1;
    //         // std::cout << t_tmp << std::endl;
    //         triangles.push_back(t_tmp);
    //     }

    //     // if(triangles.size() == 2)
    //     //     break;
    // }


    // file.close();
// exit(1);

    // /home/anonym/Schreibtisch/PhD/code/GPU_Distance_only/data/phi_SfT/real/S1/templates/template_mesh_final.obj

    Eigen::Vector3i tmp; 
    tmp << 0,1,2;
    triangles.push_back(tmp);
    tmp << 1,2,3;
    triangles.push_back(tmp);

    Eigen::Vector3d tmp1;
    tmp1 << 1,1,6;
    vertices.push_back(tmp1);
    tmp1 << 1,2,6;
    vertices.push_back(tmp1);
    tmp1 << 2,2,6;
    vertices.push_back(tmp1);
    tmp1 << 3,2,6;
    vertices.push_back(tmp1);

    std::cout <<  "number vertices " << vertices.size() << "\tnumber triangles: " << triangles.size() << std::endl;

    std::cout << "Set Camera calibration matrix\n";
    Eigen::Matrix3d K;
    K << 2,0,400,
         0,2,400,
         0,0,1;

    // Eigen::Matrix3d K;
    // K <<  971.522,0,962.134,
    //      0,944.575,554.778,
    //      0,0,1;

    optim::optGPU* opt = new optim::optGPU(10, vertices, triangles, K, true);


    std::vector<double> obs;
    int num_vertices = vertices.size();
    for(int i=0;i<num_vertices;i++) {

        Eigen::Vector3d vertex = vertices[i];

        Eigen::Vector3d tmp;
        tmp = K*vertex;

        tmp /= tmp.z();



        // obs.push_back(0);
        obs.push_back(tmp.x());
        obs.push_back(tmp.y());
        // obs.push_back(0);
        // obs.push_back(0);
        // obs.push_back(0);
    }
    
    opt->setParamater(obs, vertices.size());

    auto start = std::chrono::high_resolution_clock::now();
    
    opt->run();
    
    // hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    opt->getVertices();
    // Berechnung der verstrichenen Zeit
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    // Ausgabe der verstrichenen Zeit in Sekunden
    std::cout <<" Verstrichene Zeit: " << elapsed.count() << " Mikrosekunden" << std::endl;


    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error123: %s\n", hipGetErrorString(err));

    }



    // Test for cusolver

    // 4x4 Matrix

//     std::cout << "Test CuSolver!" << std::endl;

//     const int N=3;
//     const int nnz =3;
//     int h_csrRowPtr[N+1] = {0, 1, 2, 3}; // Zeilenstartindizes
//     int h_csrColInd[nnz] = {0, 1, 2};    // Spaltenindizes
//     double h_csrVal[nnz] = {4.0, 5.0, 6.0}; // Werte

//     double h_b[N] = {9.0, 10.0, 12.0}; // Rechte Seite b
//     double h_x[N]; // Ergebnis x
    
//      // Device-Seite Arrays
//     int *d_csrRowPtr, *d_csrColInd;
//     double *d_csrVal, *d_b, *d_x;

//     hipMalloc((void**)&d_csrRowPtr, (N+1) * sizeof(int));
//     hipMalloc((void**)&d_csrColInd, nnz * sizeof(int));
//     hipMalloc((void**)&d_csrVal, nnz * sizeof(double));
//     hipMalloc((void**)&d_b, N * sizeof(double));
//     hipMalloc((void**)&d_x, N * sizeof(double));

//     hipMemcpy(d_csrRowPtr, h_csrRowPtr, (N+1) * sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_csrColInd, h_csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_csrVal, h_csrVal, nnz * sizeof(double), hipMemcpyHostToDevice);
//     hipMemcpy(d_b, h_b, N * sizeof(double), hipMemcpyHostToDevice);

//     // cusolver handle erstellen
//     hipsolverSpHandle_t cusolverH = nullptr;
//     hipsolverStatus_t status = hipsolverSpCreate(&cusolverH);   
//     checkCusolverStatus(status);


//     int singularity = 0;

//     // CSR Cholesky info erstellen
//     // csrcholInfo_t cholInfo = nullptr;
//     // cusolverSpCreateCsrcholInfo(&cholInfo);

//     // cusparse Matrix Deskriptor erstellen
//     hipsparseMatDescr_t descrA = nullptr;
//     hipsparseCreateMatDescr(&descrA);
    
//     hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
//     hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

// double xx[3] = {0,0,0};
// double *xxx;
// xxx = &xx[0];
//     // Lösen der Gleichung Ax = b mit Cholesky-Zerlegung
//     status = hipsolverSpDcsrlsvchol(cusolverH, N, nnz, descrA, d_csrVal, d_csrRowPtr, d_csrColInd, d_b, 0, 0, d_x, &singularity);
//     checkCusolverStatus(status);


//     if (singularity > 0) {
//         std::cout << "Warnung: Die Matrix ist singulär bei index " << singularity << std::endl;
//     }

//     // Ergebnis zurück zum Host kopieren
//     hipMemcpy(h_x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);

//     // Ergebnis ausgeben
//     std::cout << "Ergebnis: ";
//     for (int i = 0; i < N; i++) {
//         std::cout << h_x[i] << " ";
//     }
//     std::cout << std::endl;

//     // Ressourcen freigeben
//     hipFree(d_csrRowPtr);
//     hipFree(d_csrColInd);
//     hipFree(d_csrVal);
//     hipFree(d_b);
//     hipFree(d_x);

//     hipsolverSpDestroy(cusolverH);

//     std::cout << "singularity: " << singularity << std::endl;
//     std::cout << "Did CuSolver" << std::endl;
     

}